#include "hip/hip_runtime.h"
#include <assert.h>
#include <matrices.h>
#include <string.h>
#include	<thrust_complex.h>
__global__ void cuDslash(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex *gamval, int *gamin,	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	char *funcname = "cuDslash";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
		for(int idirac = 0; idirac<ndirac; idirac++){
			int igork = idirac+4;
			Complex a_1, a_2;
			a_1=conj(jqq)*gamval[4*ndirac+idirac];
			//We subtract a_2, hence the minus
			a_2=-jqq*gamval[4*ndirac+idirac];
			phi[(i*ngorkov+idirac)*nc]+=a_1*r[(i*ngorkov+igork)*nc+0];
			phi[(i*ngorkov+idirac)*nc+1]+=a_1*r[(i*ngorkov+igork)*nc+1];
			phi[(i*ngorkov+igork)*nc+0]+=a_2*r[(i*ngorkov+idirac)*nc];
			phi[(i*ngorkov+igork)*nc+1]+=a_2*r[(i*ngorkov+idirac)*nc+1];
		}

		//Spacelike terms. Here's hoping I haven't put time as the zeroth component somewhere!
#ifndef NO_SPACE
		for(int mu = 0; mu <3; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int igorkov=0; igorkov<ngorkov; igorkov++){
				//FORTRAN had mod((igorkov-1),4)+1 to prevent issues with non-zero indexing in the dirac term.
				int idirac=igorkov%4;		
				int igork1 = (igorkov<4) ? gamin[mu*ndirac+idirac] : gamin[mu*ndirac+idirac]+4;
				//Can manually vectorise with a pragma?
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way
				phi[(i*ngorkov+igorkov)*nc]+=-akappa*(u11t[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc]+\
						u12t[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc+1]+\
						conj(u11t[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]-\
						u12t[did*ndim+mu]*r[(did*ngorkov+igorkov)*nc+1])+\
													  //Dirac term
													  gamval[mu*ndirac+idirac]*(u11t[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc]+\
															  u12t[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc+1]-\
															  conj(u11t[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]+\
															  u12t[did*ndim+mu]*r[(did*ngorkov+igork1)*nc+1]);

				phi[(i*ngorkov+igorkov)*nc+1]+=-akappa*(-conj(u12t[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc]+\
						conj(u11t[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc+1]+\
						conj(u12t[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]+\
						u11t[did*ndim+mu]*r[(did*ngorkov+igorkov)*nc+1])+\
														 //Dirac term
														 gamval[mu*ndirac+idirac]*(-conj(u12t[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc]+\
																 conj(u11t[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc+1]-\
																 conj(u12t[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]-\
																 u11t[did*ndim+mu]*r[(did*ngorkov+igork1)*nc+1]);
			}
		}
		//Timelike terms next. These run from igorkov=0..3 and 4..7 with slightly different rules for each
		//We can fit it into a single loop by declaring igorkovPP=igorkov+4 instead of looping igorkov=4..7  separately
		//Note that for the igorkov 4..7 loop idirac=igorkov-4, so we don't need to declare idiracPP separately
#endif
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int igorkov=0; igorkov<4; igorkov++){
			int igorkovPP=igorkov+4; 	//idirac = igorkov; It is a bit redundant but I'll mention it as that's how
			//the FORTRAN code did it.
			int igork1 = gamin[3*ndirac+igorkov];	int igork1PP = igork1+4;

			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			phi[(i*ngorkov+igorkov)*nc]+=
				-dk4p[i]*(u11t[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc]-r[(uid*ngorkov+igork1)*nc])
						+u12t[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc+1]-r[(uid*ngorkov+igork1)*nc+1]))
				-dk4m[did]*(conj(u11t[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]+r[(did*ngorkov+igork1)*nc])
						-u12t[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]+r[(did*ngorkov+igork1)*nc+1]));
			phi[(i*ngorkov+igorkov)*nc+1]+=
				-dk4p[i]*(-conj(u12t[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc]-r[(uid*ngorkov+igork1)*nc])
						+conj(u11t[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc+1]-r[(uid*ngorkov+igork1)*nc+1]))
				-dk4m[did]*(conj(u12t[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]+r[(did*ngorkov+igork1)*nc])
						+u11t[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]+r[(did*ngorkov+igork1)*nc+1]));

			//And the +4 terms. Note that dk4p and dk4m swap positions compared to the above				
			phi[(i*ngorkov+igorkovPP)*nc]+=-dk4m[i]*(u11t[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc]-r[(uid*ngorkov+igork1PP)*nc])+\
					u12t[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc+1]-r[(uid*ngorkov+igork1PP)*nc+1]))-\
													 dk4p[did]*(conj(u11t[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]+r[(did*ngorkov+igork1PP)*nc])-\
															 u12t[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]+r[(did*ngorkov+igork1PP)*nc+1]));

			phi[(i*ngorkov+igorkovPP)*nc+1]+=-dk4m[i]*(conj(-u12t[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc]-r[(uid*ngorkov+igork1PP)*nc])+\
					conj(u11t[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc+1]-r[(uid*ngorkov+igork1PP)*nc+1]))-\
														dk4p[did]*(conj(u12t[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]+r[(did*ngorkov+igork1PP)*nc])+\
																u11t[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]+r[(did*ngorkov+igork1PP)*nc+1]));
		}
#endif
	}
}
__global__ void cuDslashd(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex *gamval, int *gamin,	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	char *funcname = "cuDslashd";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
		//Diquark Term (antihermitian) The signs of a_1 and a_2 below flip under dagger
		for(int idirac = 0; idirac<ndirac; idirac++){
			int igork = idirac+4;
			Complex a_1, a_2;
			//We subtract a_1, hence the minus
			a_1=-conj(jqq)*gamval[4*ndirac+idirac];
			a_2=jqq*gamval[4*ndirac+idirac];
			phi[(i*ngorkov+idirac)*nc]+=a_1*r[(i*ngorkov+igork)*nc];
			phi[(i*ngorkov+idirac)*nc+1]+=a_1*r[(i*ngorkov+igork)*nc+1];
			phi[(i*ngorkov+igork)*nc]+=a_2*r[(i*ngorkov+idirac)*nc];
			phi[(i*ngorkov+igork)*nc+1]+=a_2*r[(i*ngorkov+idirac)*nc+1];
		}

		//Spacelike terms. Here's hoping I haven't put time as the zeroth component somewhere!
#ifndef NO_SPACE
		for(int mu = 0; mu <3; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int igorkov=0; igorkov<ngorkov; igorkov++){
				//FORTRAN had mod((igorkov-1),4)+1 to prevent issues with non-zero indexing.
				int idirac=igorkov%4;		
				int igork1 = (igorkov<4) ? gamin[mu*ndirac+idirac] : gamin[mu*ndirac+idirac]+4;
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way
				phi[(i*ngorkov+igorkov)*nc]+=
					-akappa*(      u11t[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc]
							+u12t[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc+1]
							+conj(u11t[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]
							-u12t[did*ndim+mu] *r[(did*ngorkov+igorkov)*nc+1])
					-gamval[mu*ndirac+idirac]*
					(          u11t[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc]
								  +u12t[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc+1]
								  -conj(u11t[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]
								  +u12t[did*ndim+mu] *r[(did*ngorkov+igork1)*nc+1]);

				phi[(i*ngorkov+igorkov)*nc+1]+=
					-akappa*(-conj(u12t[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc]
							+conj(u11t[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc+1]
							+conj(u12t[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]
							+u11t[did*ndim+mu] *r[(did*ngorkov+igorkov)*nc+1])
					-gamval[mu*ndirac+idirac]*
					(-conj(u12t[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc]
					 +conj(u11t[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc+1]
					 -conj(u12t[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]
					 -u11t[did*ndim+mu] *r[(did*ngorkov+igork1)*nc+1]);
			}
		}
#endif
		//Timelike terms next. These run from igorkov=0..3 and 4..7 with slightly different rules for each
		//We can fit it into a single loop by declaring igorkovPP=igorkov+4 instead of looping igorkov=4..7  separately
		//Note that for the igorkov 4..7 loop idirac=igorkov-4, so we don't need to declare idiracPP separately
		//Under dagger, dk4p and dk4m get swapped and the dirac component flips sign.
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int igorkov=0; igorkov<4; igorkov++){
			//the FORTRAN code did it.
			int igork1 = gamin[3*ndirac+igorkov];	
			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			phi[(i*ngorkov+igorkov)*nc]+=
				-dk4m[i]*(u11t[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc]+r[(uid*ngorkov+igork1)*nc])
						+u12t[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc+1]+r[(uid*ngorkov+igork1)*nc+1]))
				-dk4p[did]*(conj(u11t[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]-r[(did*ngorkov+igork1)*nc])
						-u12t[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]-r[(did*ngorkov+igork1)*nc+1]));
			phi[(i*ngorkov+igorkov)*nc+1]+=
				-dk4m[i]*(-conj(u12t[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc]+r[(uid*ngorkov+igork1)*nc])
						+conj(u11t[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc+1]+r[(uid*ngorkov+igork1)*nc+1]))
				-dk4p[did]*(conj(u12t[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]-r[(did*ngorkov+igork1)*nc])
						+u11t[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]-r[(did*ngorkov+igork1)*nc+1]));


			int igorkovPP=igorkov+4; 	//idirac = igorkov; It is a bit redundant but I'll mention it as that's how
			int igork1PP = igork1+4;
			//And the +4 terms. Note that dk4p and dk4m swap positions compared to the above				
			phi[(i*ngorkov+igorkovPP)*nc]+=-dk4p[i]*(u11t[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc]+r[(uid*ngorkov+igork1PP)*nc])+\
					u12t[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc+1]+r[(uid*ngorkov+igork1PP)*nc+1]))-\
													 dk4m[did]*(conj(u11t[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]-r[(did*ngorkov+igork1PP)*nc])-\
															 u12t[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]-r[(did*ngorkov+igork1PP)*nc+1]));

			phi[(i*ngorkov+igorkovPP)*nc+1]+=dk4p[i]*(conj(u12t[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc]+r[(uid*ngorkov+igork1PP)*nc])-\
					conj(u11t[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc+1]+r[(uid*ngorkov+igork1PP)*nc+1]))-\
														dk4m[did]*(conj(u12t[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]-r[(did*ngorkov+igork1PP)*nc])+
																u11t[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]-r[(did*ngorkov+igork1PP)*nc+1]));

		}
#endif
	}
}
__global__ void cuHdslash(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex *gamval, int *gamin,	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	char *funcname = "cuHdslash";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
#ifndef NO_SPACE
		for(int mu = 0; mu <3; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int idirac=0; idirac<ndirac; idirac++){
				//FORTRAN had mod((idirac-1),4)+1 to prevent issues with non-zero indexing.
				int igork1 = gamin[mu*ndirac+idirac];
				//Can manually vectorise with a pragma?
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way
				phi[(i*ndirac+idirac)*nc]+=-akappa*(u11t[i*ndim+mu]*r[(uid*ndirac+idirac)*nc]+\
						u12t[i*ndim+mu]*r[(uid*ndirac+idirac)*nc+1]+\
						conj(u11t[did*ndim+mu])*r[(did*ndirac+idirac)*nc]-\
						u12t[did*ndim+mu]*r[(did*ndirac+idirac)*nc+1])+\
													//Dirac term
													gamval[mu*ndirac+idirac]*(u11t[i*ndim+mu]*r[(uid*ndirac+igork1)*nc]+\
															u12t[i*ndim+mu]*r[(uid*ndirac+igork1)*nc+1]-\
															conj(u11t[did*ndim+mu])*r[(did*ndirac+igork1)*nc]+\
															u12t[did*ndim+mu]*r[(did*ndirac+igork1)*nc+1]);

				phi[(i*ndirac+idirac)*nc+1]+=-akappa*(-conj(u12t[i*ndim+mu])*r[(uid*ndirac+idirac)*nc]+\
						conj(u11t[i*ndim+mu])*r[(uid*ndirac+idirac)*nc+1]+\
						conj(u12t[did*ndim+mu])*r[(did*ndirac+idirac)*nc]+\
						u11t[did*ndim+mu]*r[(did*ndirac+idirac)*nc+1])+\
													  //Dirac term
													  gamval[mu*ndirac+idirac]*(-conj(u12t[i*ndim+mu])*r[(uid*ndirac+igork1)*nc]+\
															  conj(u11t[i*ndim+mu])*r[(uid*ndirac+igork1)*nc+1]-\
															  conj(u12t[did*ndim+mu])*r[(did*ndirac+igork1)*nc]-\
															  u11t[did*ndim+mu]*r[(did*ndirac+igork1)*nc+1]);
			}
		}
#endif
		//Timelike terms
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int idirac=0; idirac<ndirac; idirac++){
			int igork1 = gamin[3*ndirac+idirac];
			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			phi[(i*ndirac+idirac)*nc]+=
				-dk4p[i]*(u11t[i*ndim+3]*(r[(uid*ndirac+idirac)*nc]-r[(uid*ndirac+igork1)*nc])
						+u12t[i*ndim+3]*(r[(uid*ndirac+idirac)*nc+1]-r[(uid*ndirac+igork1)*nc+1]))
				-dk4m[did]*(conj(u11t[did*ndim+3])*(r[(did*ndirac+idirac)*nc]+r[(did*ndirac+igork1)*nc])
						-u12t[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]+r[(did*ndirac+igork1)*nc+1]));
			phi[(i*ndirac+idirac)*nc+1]+=
				-dk4p[i]*(-conj(u12t[i*ndim+3])*(r[(uid*ndirac+idirac)*nc]-r[(uid*ndirac+igork1)*nc])
						+conj(u11t[i*ndim+3])*(r[(uid*ndirac+idirac)*nc+1]-r[(uid*ndirac+igork1)*nc+1]))
				-dk4m[did]*(conj(u12t[did*ndim+3])*(r[(did*ndirac+idirac)*nc]+r[(did*ndirac+igork1)*nc])
						+u11t[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]+r[(did*ndirac+igork1)*nc+1]));
		}
#endif
	}
}
__global__ void cuHdslashd(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex *gamval, int *gamin,	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	char *funcname = "cuHdslashd";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
#ifndef NO_SPACE
		for(int mu = 0; mu <ndim-1; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int idirac=0; idirac<ndirac; idirac++){
				//FORTRAN had mod((idirac-1),4)+1 to prevent issues with non-zero indexing.
				int igork1 = gamin[mu*ndirac+idirac];
				//Can manually vectorise with a pragma?
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way

				phi[(i*ndirac+idirac)*nc]+=
					-akappa*(u11t[i*ndim+mu]*r[(uid*ndirac+idirac)*nc]
							+u12t[i*ndim+mu]*r[(uid*ndirac+idirac)*nc+1]
							+conj(u11t[did*ndim+mu])*r[(did*ndirac+idirac)*nc]
							-u12t[did*ndim+mu] *r[(did*ndirac+idirac)*nc+1])
					-gamval[mu*ndirac+idirac]*
					(          u11t[i*ndim+mu]*r[(uid*ndirac+igork1)*nc]
								  +u12t[i*ndim+mu]*r[(uid*ndirac+igork1)*nc+1]
								  -conj(u11t[did*ndim+mu])*r[(did*ndirac+igork1)*nc]
								  +u12t[did*ndim+mu] *r[(did*ndirac+igork1)*nc+1]);

				phi[(i*ndirac+idirac)*nc+1]+=
					-akappa*(-conj(u12t[i*ndim+mu])*r[(uid*ndirac+idirac)*nc]
							+conj(u11t[i*ndim+mu])*r[(uid*ndirac+idirac)*nc+1]
							+conj(u12t[did*ndim+mu])*r[(did*ndirac+idirac)*nc]
							+u11t[did*ndim+mu] *r[(did*ndirac+idirac)*nc+1])
					-gamval[mu*ndirac+idirac]*
					(-conj(u12t[i*ndim+mu])*r[(uid*ndirac+igork1)*nc]
					 +conj(u11t[i*ndim+mu])*r[(uid*ndirac+igork1)*nc+1]
					 -conj(u12t[did*ndim+mu])*r[(did*ndirac+igork1)*nc]
					 -u11t[did*ndim+mu] *r[(did*ndirac+igork1)*nc+1]);
			}
		}
#endif
		//Timelike terms
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int idirac=0; idirac<ndirac; idirac++){
			int igork1 = gamin[3*ndirac+idirac];
			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			//dk4m and dk4p swap under dagger
			phi[(i*ndirac+idirac)*nc]+=
				-dk4m[i]*(u11t[i*ndim+3]*(r[(uid*ndirac+idirac)*nc]+r[(uid*ndirac+igork1)*nc])
						+u12t[i*ndim+3]*(r[(uid*ndirac+idirac)*nc+1]+r[(uid*ndirac+igork1)*nc+1]))
				-dk4p[did]*(conj(u11t[did*ndim+3])*(r[(did*ndirac+idirac)*nc]-r[(did*ndirac+igork1)*nc])
						-u12t[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]-r[(did*ndirac+igork1)*nc+1]));

			phi[(i*ndirac+idirac)*nc+1]+=
				-dk4m[i]*(-conj(u12t[i*ndim+3])*(r[(uid*ndirac+idirac)*nc]+r[(uid*ndirac+igork1)*nc])
						+conj(u11t[i*ndim+3])*(r[(uid*ndirac+idirac)*nc+1]+r[(uid*ndirac+igork1)*nc+1]))
				-dk4p[did]*(conj(u12t[did*ndim+3])*(r[(did*ndirac+idirac)*nc]-r[(did*ndirac+igork1)*nc])
						+u11t[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]-r[(did*ndirac+igork1)*nc+1]));
		}
#endif
	}
}

//Float editions
__global__ void cuDslash_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu, unsigned int *id,\
		Complex_f *gamval_f,	int *gamin,	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	char *funcname = "cuDslash";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
		for(int idirac = 0; idirac<ndirac; idirac++){
			int igork = idirac+4;
			Complex_f a_1, a_2;
			a_1=conj(jqq)*gamval_f[4*ndirac+idirac];
			//We subtract a_2, hence the minus
			a_2=-jqq*gamval_f[4*ndirac+idirac];
			phi[(i*ngorkov+idirac)*nc]+=a_1*r[(i*ngorkov+igork)*nc+0];
			phi[(i*ngorkov+idirac)*nc+1]+=a_1*r[(i*ngorkov+igork)*nc+1];
			phi[(i*ngorkov+igork)*nc+0]+=a_2*r[(i*ngorkov+idirac)*nc];
			phi[(i*ngorkov+igork)*nc+1]+=a_2*r[(i*ngorkov+idirac)*nc+1];
		}

		//Spacelike terms. Here's hoping I haven't put time as the zeroth component somewhere!
#ifndef NO_SPACE
		for(int mu = 0; mu <3; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int igorkov=0; igorkov<ngorkov; igorkov++){
				//FORTRAN had mod((igorkov-1),4)+1 to prevent issues with non-zero indexing in the dirac term.
				int idirac=igorkov%4;		
				int igork1 = (igorkov<4) ? gamin[mu*ndirac+idirac] : gamin[mu*ndirac+idirac]+4;
				//Can manually vectorise with a pragma?
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way
				phi[(i*ngorkov+igorkov)*nc]+=-akappa*(u11t_f[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc]+\
						u12t_f[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc+1]+\
						conj(u11t_f[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]-\
						u12t_f[did*ndim+mu]*r[(did*ngorkov+igorkov)*nc+1])+\
													  //Dirac term
													  gamval_f[mu*ndirac+idirac]*(u11t_f[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc]+\
															  u12t_f[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc+1]-\
															  conj(u11t_f[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]+\
															  u12t_f[did*ndim+mu]*r[(did*ngorkov+igork1)*nc+1]);

				phi[(i*ngorkov+igorkov)*nc+1]+=-akappa*(-conj(u12t_f[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc]+\
						conj(u11t_f[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc+1]+\
						conj(u12t_f[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]+\
						u11t_f[did*ndim+mu]*r[(did*ngorkov+igorkov)*nc+1])+\
														 //Dirac term
														 gamval_f[mu*ndirac+idirac]*(-conj(u12t_f[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc]+\
																 conj(u11t_f[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc+1]-\
																 conj(u12t_f[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]-\
																 u11t_f[did*ndim+mu]*r[(did*ngorkov+igork1)*nc+1]);
			}
		}
		//Timelike terms next. These run from igorkov=0..3 and 4..7 with slightly different rules for each
		//We can fit it into a single loop by declaring igorkovPP=igorkov+4 instead of looping igorkov=4..7  separately
		//Note that for the igorkov 4..7 loop idirac=igorkov-4, so we don't need to declare idiracPP separately
#endif
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int igorkov=0; igorkov<4; igorkov++){
			int igorkovPP=igorkov+4; 	//idirac = igorkov; It is a bit redundant but I'll mention it as that's how
			//the FORTRAN code did it.
			int igork1 = gamin[3*ndirac+igorkov];	int igork1PP = igork1+4;

			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			phi[(i*ngorkov+igorkov)*nc]+=
				-dk4p_f[i]*(u11t_f[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc]-r[(uid*ngorkov+igork1)*nc])
						+u12t_f[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc+1]-r[(uid*ngorkov+igork1)*nc+1]))
				-dk4m_f[did]*(conj(u11t_f[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]+r[(did*ngorkov+igork1)*nc])
						-u12t_f[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]+r[(did*ngorkov+igork1)*nc+1]));
			phi[(i*ngorkov+igorkov)*nc+1]+=
				-dk4p_f[i]*(-conj(u12t_f[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc]-r[(uid*ngorkov+igork1)*nc])
						+conj(u11t_f[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc+1]-r[(uid*ngorkov+igork1)*nc+1]))
				-dk4m_f[did]*(conj(u12t_f[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]+r[(did*ngorkov+igork1)*nc])
						+u11t_f[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]+r[(did*ngorkov+igork1)*nc+1]));

			//And the +4 terms. Note that dk4p_f and dk4m_f swap positions compared to the above				
			phi[(i*ngorkov+igorkovPP)*nc]+=-dk4m_f[i]*(u11t_f[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc]-r[(uid*ngorkov+igork1PP)*nc])+\
					u12t_f[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc+1]-r[(uid*ngorkov+igork1PP)*nc+1]))-\
													 dk4p_f[did]*(conj(u11t_f[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]+r[(did*ngorkov+igork1PP)*nc])-\
															 u12t_f[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]+r[(did*ngorkov+igork1PP)*nc+1]));

			phi[(i*ngorkov+igorkovPP)*nc+1]+=-dk4m_f[i]*(conj(-u12t_f[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc]-r[(uid*ngorkov+igork1PP)*nc])+\
					conj(u11t_f[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc+1]-r[(uid*ngorkov+igork1PP)*nc+1]))-\
														dk4p_f[did]*(conj(u12t_f[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]+r[(did*ngorkov+igork1PP)*nc])+\
																u11t_f[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]+r[(did*ngorkov+igork1PP)*nc+1]));
		}
#endif
	}
}
__global__ void cuDslashd_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu, unsigned int *id,\
		Complex_f *gamval_f,	int *gamin,	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	char *funcname = "cuDslashd";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
		//Diquark Term (antihermitian) The signs of a_1 and a_2 below flip under dagger
		for(int idirac = 0; idirac<ndirac; idirac++){
			int igork = idirac+4;
			Complex_f a_1, a_2;
			//We subtract a_1, hence the minus
			a_1=-conj(jqq)*gamval_f[4*ndirac+idirac];
			a_2=jqq*gamval_f[4*ndirac+idirac];
			phi[(i*ngorkov+idirac)*nc]+=a_1*r[(i*ngorkov+igork)*nc];
			phi[(i*ngorkov+idirac)*nc+1]+=a_1*r[(i*ngorkov+igork)*nc+1];
			phi[(i*ngorkov+igork)*nc]+=a_2*r[(i*ngorkov+idirac)*nc];
			phi[(i*ngorkov+igork)*nc+1]+=a_2*r[(i*ngorkov+idirac)*nc+1];
		}

		//Spacelike terms. Here's hoping I haven't put time as the zeroth component somewhere!
#ifndef NO_SPACE
		for(int mu = 0; mu <3; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int igorkov=0; igorkov<ngorkov; igorkov++){
				//FORTRAN had mod((igorkov-1),4)+1 to prevent issues with non-zero indexing.
				int idirac=igorkov%4;		
				int igork1 = (igorkov<4) ? gamin[mu*ndirac+idirac] : gamin[mu*ndirac+idirac]+4;
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way
				phi[(i*ngorkov+igorkov)*nc]+=
					-akappa*(u11t_f[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc]
							+u12t_f[i*ndim+mu]*r[(uid*ngorkov+igorkov)*nc+1]
							+conj(u11t_f[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]
							-u12t_f[did*ndim+mu] *r[(did*ngorkov+igorkov)*nc+1])
					-gamval_f[mu*ndirac+idirac]*
					(u11t_f[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc]
					 +u12t_f[i*ndim+mu]*r[(uid*ngorkov+igork1)*nc+1]
					 -conj(u11t_f[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]
					 +u12t_f[did*ndim+mu] *r[(did*ngorkov+igork1)*nc+1]);

				phi[(i*ngorkov+igorkov)*nc+1]+=
					-akappa*(-conj(u12t_f[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc]
							+conj(u11t_f[i*ndim+mu])*r[(uid*ngorkov+igorkov)*nc+1]
							+conj(u12t_f[did*ndim+mu])*r[(did*ngorkov+igorkov)*nc]
							+u11t_f[did*ndim+mu] *r[(did*ngorkov+igorkov)*nc+1])
					-gamval_f[mu*ndirac+idirac]*
					(-conj(u12t_f[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc]
					 +conj(u11t_f[i*ndim+mu])*r[(uid*ngorkov+igork1)*nc+1]
					 -conj(u12t_f[did*ndim+mu])*r[(did*ngorkov+igork1)*nc]
					 -u11t_f[did*ndim+mu] *r[(did*ngorkov+igork1)*nc+1]);
			}
		}
#endif
		//Timelike terms next. These run from igorkov=0..3 and 4..7 with slightly different rules for each
		//We can fit it into a single loop by declaring igorkovPP=igorkov+4 instead of looping igorkov=4..7  separately
		//Note that for the igorkov 4..7 loop idirac=igorkov-4, so we don't need to declare idiracPP separately
		//Under dagger, dk4p_f and dk4m_f get swapped and the dirac component flips sign.
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int igorkov=0; igorkov<4; igorkov++){
			//the FORTRAN code did it.
			int igork1 = gamin[3*ndirac+igorkov];	
			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			phi[(i*ngorkov+igorkov)*nc]+=
				-dk4m_f[i]*(u11t_f[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc]+r[(uid*ngorkov+igork1)*nc])
						+u12t_f[i*ndim+3]*(r[(uid*ngorkov+igorkov)*nc+1]+r[(uid*ngorkov+igork1)*nc+1]))
				-dk4p_f[did]*(conj(u11t_f[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]-r[(did*ngorkov+igork1)*nc])
						-u12t_f[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]-r[(did*ngorkov+igork1)*nc+1]));
			phi[(i*ngorkov+igorkov)*nc+1]+=
				-dk4m_f[i]*(-conj(u12t_f[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc]+r[(uid*ngorkov+igork1)*nc])
						+conj(u11t_f[i*ndim+3])*(r[(uid*ngorkov+igorkov)*nc+1]+r[(uid*ngorkov+igork1)*nc+1]))
				-dk4p_f[did]*(conj(u12t_f[did*ndim+3])*(r[(did*ngorkov+igorkov)*nc]-r[(did*ngorkov+igork1)*nc])
						+u11t_f[did*ndim+3] *(r[(did*ngorkov+igorkov)*nc+1]-r[(did*ngorkov+igork1)*nc+1]));


			int igorkovPP=igorkov+4; 	//idirac = igorkov; It is a bit redundant but I'll mention it as that's how
			int igork1PP = igork1+4;
			//And the +4 terms. Note that dk4p_f and dk4m_f swap positions compared to the above				
			phi[(i*ngorkov+igorkovPP)*nc]+=-dk4p_f[i]*(u11t_f[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc]+r[(uid*ngorkov+igork1PP)*nc])+\
					u12t_f[i*ndim+3]*(r[(uid*ngorkov+igorkovPP)*nc+1]+r[(uid*ngorkov+igork1PP)*nc+1]))-\
													 dk4m_f[did]*(conj(u11t_f[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]-r[(did*ngorkov+igork1PP)*nc])-\
															 u12t_f[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]-r[(did*ngorkov+igork1PP)*nc+1]));

			phi[(i*ngorkov+igorkovPP)*nc+1]+=dk4p_f[i]*(conj(u12t_f[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc]+r[(uid*ngorkov+igork1PP)*nc])-\
					conj(u11t_f[i*ndim+3])*(r[(uid*ngorkov+igorkovPP)*nc+1]+r[(uid*ngorkov+igork1PP)*nc+1]))-\
														dk4m_f[did]*(conj(u12t_f[did*ndim+3])*(r[(did*ngorkov+igorkovPP)*nc]-r[(did*ngorkov+igork1PP)*nc])+
																u11t_f[did*ndim+3]*(r[(did*ngorkov+igorkovPP)*nc+1]-r[(did*ngorkov+igork1PP)*nc+1]));

		}
#endif
	}
}
__global__ void cuHdslash_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu, unsigned int *id,\
		Complex_f *gamval_f,	int *gamin,	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	char *funcname = "cuHdslash";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
#ifndef NO_SPACE
		for(int mu = 0; mu <3; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int idirac=0; idirac<ndirac; idirac++){
				//FORTRAN had mod((idirac-1),4)+1 to prevent issues with non-zero indexing.
				int igork1 = gamin[mu*ndirac+idirac];
				//Can manually vectorise with a pragma?
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way
				phi[(i*ndirac+idirac)*nc]+=-akappa*(u11t_f[i*ndim+mu]*r[(uid*ndirac+idirac)*nc]+\
						u12t_f[i*ndim+mu]*r[(uid*ndirac+idirac)*nc+1]+\
						conj(u11t_f[did*ndim+mu])*r[(did*ndirac+idirac)*nc]-\
						u12t_f[did*ndim+mu]*r[(did*ndirac+idirac)*nc+1])+\
													//Dirac term
													gamval_f[mu*ndirac+idirac]*(u11t_f[i*ndim+mu]*r[(uid*ndirac+igork1)*nc]+\
															u12t_f[i*ndim+mu]*r[(uid*ndirac+igork1)*nc+1]-\
															conj(u11t_f[did*ndim+mu])*r[(did*ndirac+igork1)*nc]+\
															u12t_f[did*ndim+mu]*r[(did*ndirac+igork1)*nc+1]);

				phi[(i*ndirac+idirac)*nc+1]+=-akappa*(-conj(u12t_f[i*ndim+mu])*r[(uid*ndirac+idirac)*nc]+\
						conj(u11t_f[i*ndim+mu])*r[(uid*ndirac+idirac)*nc+1]+\
						conj(u12t_f[did*ndim+mu])*r[(did*ndirac+idirac)*nc]+\
						u11t_f[did*ndim+mu]*r[(did*ndirac+idirac)*nc+1])+\
													  //Dirac term
													  gamval_f[mu*ndirac+idirac]*(-conj(u12t_f[i*ndim+mu])*r[(uid*ndirac+igork1)*nc]+\
															  conj(u11t_f[i*ndim+mu])*r[(uid*ndirac+igork1)*nc+1]-\
															  conj(u12t_f[did*ndim+mu])*r[(did*ndirac+igork1)*nc]-\
															  u11t_f[did*ndim+mu]*r[(did*ndirac+igork1)*nc+1]);
			}
		}
#endif
		//Timelike terms
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int idirac=0; idirac<ndirac; idirac++){
			int igork1 = gamin[3*ndirac+idirac];
			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			phi[(i*ndirac+idirac)*nc]+=
				-dk4p_f[i]*(u11t_f[i*ndim+3]*(r[(uid*ndirac+idirac)*nc]-r[(uid*ndirac+igork1)*nc])
						+u12t_f[i*ndim+3]*(r[(uid*ndirac+idirac)*nc+1]-r[(uid*ndirac+igork1)*nc+1]))
				-dk4m_f[did]*(conj(u11t_f[did*ndim+3])*(r[(did*ndirac+idirac)*nc]+r[(did*ndirac+igork1)*nc])
						-u12t_f[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]+r[(did*ndirac+igork1)*nc+1]));
			phi[(i*ndirac+idirac)*nc+1]+=
				-dk4p_f[i]*(-conj(u12t_f[i*ndim+3])*(r[(uid*ndirac+idirac)*nc]-r[(uid*ndirac+igork1)*nc])
						+conj(u11t_f[i*ndim+3])*(r[(uid*ndirac+idirac)*nc+1]-r[(uid*ndirac+igork1)*nc+1]))
				-dk4m_f[did]*(conj(u12t_f[did*ndim+3])*(r[(did*ndirac+idirac)*nc]+r[(did*ndirac+igork1)*nc])
						+u11t_f[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]+r[(did*ndirac+igork1)*nc+1]));
		}
#endif
	}
}
__global__ void cuHdslashd_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu, unsigned int *id,\
		Complex_f *gamval_f,	int *gamin,	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	char *funcname = "cuHdslashd";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
#ifndef NO_SPACE
		for(int mu = 0; mu <ndim-1; mu++){
			int did=id[mu+ndim*i]; int uid = iu[mu+ndim*i];
			for(int idirac=0; idirac<ndirac; idirac++){
				//FORTRAN had mod((idirac-1),4)+1 to prevent issues with non-zero indexing.
				int igork1 = gamin[mu*ndirac+idirac];
				//Can manually vectorise with a pragma?
				//Wilson + Dirac term in that order. Definitely easier
				//to read when split into different loops, but should be faster this way

				phi[(i*ndirac+idirac)*nc]+=
					-akappa*(u11t_f[i*ndim+mu]*r[(uid*ndirac+idirac)*nc]
							+u12t_f[i*ndim+mu]*r[(uid*ndirac+idirac)*nc+1]
							+conj(u11t_f[did*ndim+mu])*r[(did*ndirac+idirac)*nc]
							-u12t_f[did*ndim+mu] *r[(did*ndirac+idirac)*nc+1])
					-gamval_f[mu*ndirac+idirac]*
					(u11t_f[i*ndim+mu]*r[(uid*ndirac+igork1)*nc]
					 +u12t_f[i*ndim+mu]*r[(uid*ndirac+igork1)*nc+1]
					 -conj(u11t_f[did*ndim+mu])*r[(did*ndirac+igork1)*nc]
					 +u12t_f[did*ndim+mu] *r[(did*ndirac+igork1)*nc+1]);

				phi[(i*ndirac+idirac)*nc+1]+=
					-akappa*(-conj(u12t_f[i*ndim+mu])*r[(uid*ndirac+idirac)*nc]
							+conj(u11t_f[i*ndim+mu])*r[(uid*ndirac+idirac)*nc+1]
							+conj(u12t_f[did*ndim+mu])*r[(did*ndirac+idirac)*nc]
							+u11t_f[did*ndim+mu] *r[(did*ndirac+idirac)*nc+1])
					-gamval_f[mu*ndirac+idirac]*
					(-conj(u12t_f[i*ndim+mu])*r[(uid*ndirac+igork1)*nc]
					 +conj(u11t_f[i*ndim+mu])*r[(uid*ndirac+igork1)*nc+1]
					 -conj(u12t_f[did*ndim+mu])*r[(did*ndirac+igork1)*nc]
					 -u11t_f[did*ndim+mu] *r[(did*ndirac+igork1)*nc+1]);
			}
		}
#endif
		//Timelike terms
		int did=id[3+ndim*i]; int uid = iu[3+ndim*i];
#ifndef NO_TIME
		for(int idirac=0; idirac<ndirac; idirac++){
			int igork1 = gamin[3*ndirac+idirac];
			//Factorising for performance, we get dk4?*u1?*(+/-r_wilson -/+ r_dirac)
			//dk4m_f and dk4p_f swap under dagger
			phi[(i*ndirac+idirac)*nc]+=
				-dk4m_f[i]*(u11t_f[i*ndim+3]*(r[(uid*ndirac+idirac)*nc]+r[(uid*ndirac+igork1)*nc])
						+u12t_f[i*ndim+3]*(r[(uid*ndirac+idirac)*nc+1]+r[(uid*ndirac+igork1)*nc+1]))
				-dk4p_f[did]*(conj(u11t_f[did*ndim+3])*(r[(did*ndirac+idirac)*nc]-r[(did*ndirac+igork1)*nc])
						-u12t_f[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]-r[(did*ndirac+igork1)*nc+1]));

			phi[(i*ndirac+idirac)*nc+1]+=
				-dk4m_f[i]*(-conj(u12t_f[i*ndim+3])*(r[(uid*ndirac+idirac)*nc]+r[(uid*ndirac+igork1)*nc])
						+conj(u11t_f[i*ndim+3])*(r[(uid*ndirac+idirac)*nc+1]+r[(uid*ndirac+igork1)*nc+1]))
				-dk4p_f[did]*(conj(u12t_f[did*ndim+3])*(r[(did*ndirac+idirac)*nc]-r[(did*ndirac+igork1)*nc])
						+u11t_f[did*ndim+3] *(r[(did*ndirac+idirac)*nc+1]-r[(did*ndirac+igork1)*nc+1]));
		}
#endif
	}
}

__global__ void cuNew_trial(double dt, double *pp, Complex *u11t, Complex *u12t){
	char *funcname = "New_trial";
	const	int gsize = gridDim.x*gridDim.y*gridDim.z;
	const	int bsize = blockDim.x*blockDim.y*blockDim.z;
	const	int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const	int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
		for(int mu = 0; mu<ndim; mu++){
			//Sticking to what was in the FORTRAN for variable names.
			//CCC for cosine SSS for sine AAA for...
			//Re-exponentiating the force field. Can be done analytically in SU(2)
			//using sine and cosine which is nice
			double AAA = dt*sqrt(pp[i*nadj*ndim+mu]*pp[i*nadj*ndim+mu]\
					+pp[(i*nadj+1)*ndim+mu]*pp[(i*nadj+1)*ndim+mu]\
					+pp[(i*nadj+2)*ndim+mu]*pp[(i*nadj+2)*ndim+mu]);
			double CCC = cos(AAA);
			double SSS = dt*sin(AAA)/AAA;
			Complex a11 = CCC+I*SSS*pp[(i*nadj+2)*ndim+mu];
			Complex a12 = pp[(i*nadj+1)*ndim+mu]*SSS + I*SSS*pp[i*nadj*ndim+mu];
			//b11 and b12 are u11t and u12t terms, so we'll use u12t directly
			//but use b11 for u11t to prevent RAW dependency
			Complex b11 = u11t[i*ndim+mu];
			u11t[i*ndim+mu] = a11*b11-a12*conj(u12t[i*ndim+mu]);
			u12t[i*ndim+mu] = a11*u12t[i*ndim+mu]+a12*conj(b11);
		}
	}
}
__global__ void cuReunitarise(Complex *u11t, Complex * u12t){
	/*
	 * Reunitarises u11t and u12t as in conj(u11t[i])*u11t[i]+conj(u12t[i])*u12t[i]=1
	 *
	 * If you're looking at the FORTRAN code be careful. There are two header files
	 * for the /trial/ header. One with u11 u12 (which was included here originally)
	 * and the other with u11t and u12t.
	 *
	 * Globals:
	 * =======
	 * u11t, u12t
	 *
	 * Returns:
	 * ========
	 * Zero on success, integer error code otherwise
	 */
	const char *funcname = "Reunitarise";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId; i<kvol*ndim; i+=gsize){
		//Declaring anorm inside the loop will hopefully let the compiler know it
		//is safe to vectorise aggessively
		double anorm=sqrt(conj(u11t[i])*u11t[i]+conj(u12t[i])*u12t[i]).real();
		//		Exception handling code. May be faster to leave out as the exit prevents vectorisation.
		//		if(anorm==0){
		//			fprintf(stderr, "Error %i in %s on rank %i: anorm = 0 for μ=%i and i=%i.\nExiting...\n\n",
		//					DIVZERO, funcname, rank, mu, i);
		//			MPI_Finalise();
		//			exit(DIVZERO);
		//		}
		u11t[i]/=anorm;
		u12t[i]/=anorm;
	}
}

int Dslash(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex gamval[5][4], int gamin[4][4],	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parametrer:
	 * ==========
	 *
	 * Complex *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Dslash";
	//Get the halos in order
	ZHalo_swap_all(r, 16);

	//Mass term
	memcpy(phi, r, kferm*sizeof(Complex));
	//	hipMemPrefetchAsync(u11t,kvol+halo,0
	cuDslash<<<dimGrid,dimBlock>>>(phi,r,u11t,u12t,iu,id,&gamval[0][0],&gamin[0][0],dk4m,dk4p,jqq,akappa);
	return 0;
}
int Dslashd(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex gamval[5][4], int gamin[4][4],	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parameter:
	 * ==========
	 *
	 * Complex *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Dslashd";
	//Get the halos in order
	ZHalo_swap_all(r, 16);

	//Mass term
	memcpy(phi, r, kferm*sizeof(Complex));
	//	hipMemPrefetchAsync(u11t,kvol+halo,0
	cuDslashd<<<dimGrid,dimBlock>>>(phi,r,u11t,u12t,iu,id,&gamval[0][0],&gamin[0][0],dk4m,dk4p,jqq,akappa);
	return 0;
}
int Hdslash(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex gamval[5][4], int gamin[4][4],	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parametrer:
	 * ==========
	 *
	 * Complex *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Hdslash";
	//Get the halos in order
	ZHalo_swap_all(r, 8);

	//Mass term
	memcpy(phi, r, kferm2*sizeof(Complex));
	//Spacelike term
		//	hipMemPrefetchAsync(u11t,kvol+halo,0
	cuHdslash<<<dimGrid,dimBlock>>>(phi,r,u11t,u12t,iu,id,&gamval[0][0],&gamin[0][0],dk4m,dk4p,jqq,akappa);
	return 0;
}
int Hdslashd(Complex *phi, Complex *r, Complex *u11t, Complex *u12t,unsigned int *iu,unsigned  int *id,\
		Complex gamval[5][4], int gamin[4][4],	double *dk4m, double *dk4p, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parametrer:
	 * ==========
	 *
	 * Complex *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Hdslashd";
	//Get the halos in order. Because C is row major, we need to extract the correct
	//terms for each halo first. Changing the indices was considered but that caused
	//issues with the BLAS routines.
	ZHalo_swap_all(r, 8);

	//Looks like flipping the array ordering for C has meant a lot
	//of for loops. Sense we're jumping around quite a bit the cache is probably getting refreshed
	//anyways so memory access patterns mightn't be as big of an limiting factor here anyway

	//Mass term
	memcpy(phi, r, kferm2*sizeof(Complex));
	//Spacelike term
	cuHdslashd<<<dimGrid,dimBlock>>>(phi,r,u11t,u12t,iu,id,&gamval[0][0],&gamin[0][0],dk4m,dk4p,jqq,akappa);
	return 0;
}

inline int Reunitarise(Complex *u11t, Complex *u12t){
	cuReunitarise<<<dimGrid,dimBlock>>>(u11t,u12t);
	return 0;
}
int New_trial(double dt, double *pp, Complex *u11t, Complex *u12t){
	cuNew_trial<<<dimGrid,dimBlock>>>(dt,pp,u11t,u12t);
	return 0;
}
//Float editions
int Dslash_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu, unsigned int *id,\
		Complex_f gamval_f[5][4],	int gamin[4][4],	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parametrer:
	 * ==========
	 *
	 * Complex *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Dslash_f";
	//Get the halos in order
	CHalo_swap_all(r, 16);

	//Mass term
	memcpy(phi, r, kferm*sizeof(Complex_f));
	//	hipMemPrefetchAsync(u11t,kvol+halo,0
	cuDslash_f<<<dimGrid,dimBlock>>>(phi,r,u11t_f,u12t_f,iu,id,&gamval_f[0][0],&gamin[0][0],dk4m_f,dk4p_f,jqq,akappa);
	return 0;
}
int Dslashd_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu,unsigned int *id,\
		Complex_f gamval_f[5][4],		int gamin[4][4],	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parameter:
	 * ==========
	 *
	 * Complex *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Dslashd_f";
	//Get the halos in order
	CHalo_swap_all(r, 16);

	//Mass term
	memcpy(phi, r, kferm*sizeof(Complex_f));
	//	hipMemPrefetchAsync(u11t,kvol+halo,0
	cuDslashd_f<<<dimGrid,dimBlock>>>(phi,r,u11t_f,u12t_f,iu,id,&gamval_f[0][0],&gamin[0][0],dk4m_f,dk4p_f,jqq,akappa);
	return 0;
}
int Hdslash_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned  int *iu,unsigned  int *id,\
		Complex_f gamval_f[5][4],	int gamin[4][4],	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parametrer:
	 * ==========
	 *
	 * Complex_f *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex_f r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Hdslash_f";
	//Get the halos in order
	CHalo_swap_all(r, 8);

	//Mass term
	memcpy(phi, r, kferm2*sizeof(Complex_f));
	cuHdslash_f<<<dimGrid,dimBlock>>>(phi,r,u11t_f,u12t_f,iu,id,&gamval_f[0][0],&gamin[0][0],dk4m_f,dk4p_f,jqq,akappa);
	return 0;
}
int Hdslashd_f(Complex_f *phi, Complex_f *r, Complex_f *u11t_f, Complex_f *u12t_f,unsigned int *iu,unsigned int *id,\
		Complex_f gamval_f[5][4],int gamin[4][4],	float *dk4m_f, float *dk4p_f, Complex_f jqq, float akappa){
	/*
	 * Evaluates phi= M*r
	 *
	 * Globals
	 * =======
	 * u11t, u12t, dk4p, dk4m, (*akappa_d), jqq_d 
	 *
	 * Calls:
	 * ======
	 * zhaloswapdir, chaloswapdir, zhaloswapall (Non-mpi version could do without these)
	 *
	 * Parametrer:
	 * ==========
	 *
	 * Complex_f *phi:	The result container. This is NOT THE SAME AS THE GLOBAL Phi. But
	 * 			for consistency with the fortran code I'll keep the name here
	 * Complex_f r:		The array being acted on by M
	 *
	 * Returns:
	 * Zero on success, integer error code otherwise
	 */
	char *funcname = "Hdslashd_f";
	//Get the halos in order. Because C is row major, we need to extract the correct
	//terms for each halo first. Changing the indices was considered but that caused
	//issues with the BLAS routines.
	CHalo_swap_all(r, 8);

	//Looks like flipping the array ordering for C has meant a lot
	//of for loops. Sense we're jumping around quite a bit the cache is probably getting refreshed
	//anyways so memory access patterns mightn't be as big of an limiting factor here anyway

	//Mass term
	memcpy(phi, r, kferm2*sizeof(Complex_f));
	cuHdslashd_f<<<dimGrid,dimBlock>>>(phi,r,u11t_f,u12t_f,iu,id,&gamval_f[0][0],&gamin[0][0],dk4m_f,dk4p_f,jqq,akappa);
	return 0;
}

#ifdef DIAGNOSTIC
int Diagnostics(int istart){
	/*
	 * Routine to check if the multiplication routines are working or not
	 * How I hope this will work is that
	 * 1)	Initialise the system
	 * 2) Just after the initialisation of the system but before anything
	 * 	else call this routine using the C Preprocessor.
	 * 3) Give dummy values for the fields and then do work with them
	 * Caveats? Well this could get messy if we call something we didn't
	 * realise was being called and hadn't initialised it properly (Congradq
	 * springs to mind straight away)
	 */
	char *funcname = "Diagnostics";

	//Initialise the arrays being used. Just going to assume MKL is being
	//used here will also assert the number of flavours for now to avoid issues
	//later
	assert(nf==1);

#ifdef __NVCC__
	int device=-1;
	hipGetDevice(&device);
	hipMallocManaged(&R1, kfermHalo*sizeof(Complex),hipMemAttachGlobal);
	hipMallocManaged(&xi, kfermHalo*sizeof(Complex),hipMemAttachGlobal);
	hipMallocManaged(&Phi, nf*kfermHalo*sizeof(Complex),hipMemAttachGlobal);
	hipMallocManaged(&X0, nf*kfermHalo*sizeof(Complex),hipMemAttachGlobal);
	hipMallocManaged(&X1, kferm2Halo*sizeof(Complex),hipMemAttachGlobal);
	hipMallocManaged(&pp, kmomHalo*sizeof(double),hipMemAttachGlobal);
	double *dSdpi;
	hipMallocManaged(&dSdpi, kmomHalo*sizeof(double),hipMemAttachGlobal);
#elif defined USE_MKL
	R1= mkl_malloc(kfermHalo*sizeof(complex),AVX);
	xi= mkl_malloc(kfermHalo*sizeof(complex),AVX);
	Phi= mkl_malloc(nf*kfermHalo*sizeof(complex),AVX); 
	X0= mkl_malloc(nf*kferm2Halo*sizeof(complex),AVX); 
	X1= mkl_malloc(kferm2Halo*sizeof(complex),AVX); 
	double *dSdpi = mkl_malloc(kmomHalo*sizeof(double), AVX);
#else
	R1= aligned_alloc(AVX,kfermHalo*sizeof(Complex));
	xi= aligned_alloc(AVX,kfermHalo*sizeof(Complex));
	Phi= aligned_alloc(AVX,nf*kfermHalo*sizeof(Complex)); 
	X0= aligned_alloc(AVX,nf*kferm2Halo*sizeof(Complex)); 
	X1= aligned_alloc(AVX,kferm2Halo*sizeof(Complex)); 
	pp = aligned_alloc(AVX,kmomHalo*sizeof(double));
	double *dSdpi = aligned_alloc(AVX,kmomHalo*sizeof(double));
#endif
	//pp is the momentum field

	//Trial fields don't get modified so I'll set them up outside
	switch(istart){
		case(2):
#pragma omp parallel for simd aligned(u11t:AVX,u12t:AVX)
			for(int i =0; i<ndim*kvol; i+=8){
				u11t[i+0]=1.0+I; u12t[i]=1.0+I;
				u11t[i+1]=1.0-I; u12t[i+1]=1.0+I;
				u11t[i+2]=1.0+I; u12t[i+2]=1.0-I;
				u11t[i+3]=1.0-I; u12t[i+3]=1.0-I;
				u11t[i+4]=-1.0+I; u12t[i+4]=1.0+I;
				u11t[i+5]=1.0+I; u12t[i+5]=-1.0+I;
				u11t[i+6]=-1.0+I; u12t[i+6]=-1.0+I;
				u11t[i+7]=-1.0-I; u12t[i+7]=-1.0-I;
			}
			break;
		case(1):
			Trial_Exchange();
#pragma omp parallel sections num_threads(2)
			{
#pragma omp section
				{
					FILE *trial_out = fopen("u11t", "w");
					for(int i=0;i<ndim*(kvol+halo);i+=4)
						fprintf(trial_out,"%f+%fI\t%f+%fI\t%f+%fI\t%f+%fI\n",
								creal(u11t[i]),cimag(u11t[i]),creal(u11t[i+1]),cimag(u11t[i+1]),
								creal(u11t[2+i]),cimag(u11t[2+i]),creal(u11t[i+3]),cimag(u11t[i+3]));

					fclose(trial_out);
				}
#pragma omp section
				{
					FILE *trial_out = fopen("u12t", "w");
					for(int i=0;i<ndim*(kvol+halo);i+=4)
						fprintf(trial_out,"%f+%fI\t%f+%fI\t%f+%fI\t%f+%fI\n",
								creal(u12t[i]),cimag(u12t[i]),creal(u12t[i+1]),cimag(u12t[i+1]),
								creal(u12t[2+i]),cimag(u12t[2+i]),creal(u12t[i+3]),cimag(u12t[i+3]));
					fclose(trial_out);
				}
			}
			break;
		default:
			//Cold start as a default
			memcpy(u11,u11t,kvol*ndim*sizeof(Complex));
			memcpy(u12t,u12t,kvol*ndim*sizeof(Complex));
			break;
	}
#pragma omp parallel for simd aligned(u11t:AVX,u12t:AVX) 
	for(int i=0; i<kvol*ndim; i++){
		//Declaring anorm inside the loop will hopefully let the compiler know it
		//is safe to vectorise aggessively
		double anorm;
		anorm=creal(sqrt(conj(u11t[i])*u11t[i]+conj(u12t[i])*u12t[i]));
		assert(anorm!=0);
		u11t[i]/=anorm;
		u12t[i]/=anorm;
	}

	Trial_Exchange();
	for(int test = 0; test<=6; test++){
		//Reset between tests
#pragma omp parallel for simd
		for(int i=0; i<kferm; i++){
			R1[i]=0.5; Phi[i]=0.5;xi[i]=0.5;
		}
#pragma omp parallel for simd
		for(int i=0; i<kferm2; i++){
			X0[i]=0.5;
			X1[i]=0.5;
		}
#pragma omp parallel for simd
		for(int i=0; i<kmomHalo; i++)
			dSdpi[i] = 0;
		FILE *output_old, *output;
		switch(test){
			case(0):
				output_old = fopen("dslash_old", "w");
				for(int i = 0; i< kferm; i+=8)
					fprintf(output_old, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(xi[i]),cimag(xi[i]),creal(xi[i+1]),cimag(xi[i+1]),
							creal(xi[i+2]),cimag(xi[i+2]),creal(xi[i+3]),cimag(xi[i+3]),
							creal(xi[i+4]),cimag(xi[i+4]),creal(xi[i+5]),cimag(xi[i+5]),
							creal(xi[i+6]),cimag(xi[i+6]),creal(xi[i+7]),cimag(xi[i+7])	);
				fclose(output_old);
				Dslash(xi, R1);
				output = fopen("dslash", "w");
				for(int i = 0; i< kferm; i+=8)
					fprintf(output, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(xi[i]),cimag(xi[i]),creal(xi[i+1]),cimag(xi[i+1]),
							creal(xi[i+2]),cimag(xi[i+2]),creal(xi[i+3]),cimag(xi[i+3]),
							creal(xi[i+4]),cimag(xi[i+4]),creal(xi[i+5]),cimag(xi[i+5]),
							creal(xi[i+6]),cimag(xi[i+6]),creal(xi[i+7]),cimag(xi[i+7])	);
				fclose(output);
				break;
			case(1):
				output_old = fopen("dslashd_old", "w");
				for(int i = 0; i< kferm; i+=8)
					fprintf(output_old, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(xi[i]),cimag(xi[i]),creal(xi[i+1]),cimag(xi[i+1]),
							creal(xi[i+2]),cimag(xi[i+2]),creal(xi[i+3]),cimag(xi[i+3]),
							creal(xi[i+4]),cimag(xi[i+4]),creal(xi[i+5]),cimag(xi[i+5]),
							creal(xi[i+6]),cimag(xi[i+6]),creal(xi[i+7]),cimag(xi[i+7])	);
				fclose(output_old);
				Dslashd(xi, R1);
				output = fopen("dslashd", "w");
				for(int i = 0; i< kferm; i+=8)
					fprintf(output, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(xi[i]),cimag(xi[i]),creal(xi[i+1]),cimag(xi[i+1]),
							creal(xi[i+2]),cimag(xi[i+2]),creal(xi[i+3]),cimag(xi[i+3]),
							creal(xi[i+4]),cimag(xi[i+4]),creal(xi[i+5]),cimag(xi[i+5]),
							creal(xi[i+6]),cimag(xi[i+6]),creal(xi[i+7]),cimag(xi[i+7])	);
				fclose(output);
				break;
			case(2):	
				output_old = fopen("hdslash_old", "w");
				for(int i = 0; i< kferm2; i+=8)
					fprintf(output_old, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(X1[i]),cimag(X1[i]),creal(X1[i+1]),cimag(X1[i+1]),
							creal(X1[i+2]),cimag(X1[i+2]),creal(X1[i+3]),cimag(X1[i+3]),
							creal(X1[i+4]),cimag(X1[i+4]),creal(X1[i+5]),cimag(X1[i+5]),
							creal(X1[i+6]),cimag(X1[i+6]),creal(X1[i+7]),cimag(X1[i+7]));
				fclose(output_old);
				Hdslash(X1, X0);
				output = fopen("hdslash", "w");
				for(int i = 0; i< kferm2; i+=8)
					fprintf(output, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(X1[i]),cimag(X1[i]),creal(X1[i+1]),cimag(X1[i+1]),
							creal(X1[i+2]),cimag(X1[i+2]),creal(X1[i+3]),cimag(X1[i+3]),
							creal(X1[i+4]),cimag(X1[i+4]),creal(X1[i+5]),cimag(X1[i+5]),
							creal(X1[i+6]),cimag(X1[i+6]),creal(X1[i+7]),cimag(X1[i+7]));
				fclose(output);
				break;
			case(3):	
				output_old = fopen("hdslashd_old", "w");
				for(int i = 0; i< kferm2; i+=8)
					fprintf(output_old, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(X1[i]),cimag(X1[i]),creal(X1[i+1]),cimag(X1[i+1]),
							creal(X1[i+2]),cimag(X1[i+2]),creal(X1[i+3]),cimag(X1[i+3]),
							creal(X1[i+4]),cimag(X1[i+4]),creal(X1[i+5]),cimag(X1[i+5]),
							creal(X1[i+6]),cimag(X1[i+6]),creal(X1[i+7]),cimag(X1[i+7]));
				fclose(output_old);
				Hdslashd(X1, X0);
				output = fopen("hdslashd", "w");
				for(int i = 0; i< kferm2; i+=8)
					fprintf(output, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(X1[i]),cimag(X1[i]),creal(X1[i+1]),cimag(X1[i+1]),
							creal(X1[i+2]),cimag(X1[i+2]),creal(X1[i+3]),cimag(X1[i+3]),
							creal(X1[i+4]),cimag(X1[i+4]),creal(X1[i+5]),cimag(X1[i+5]),
							creal(X1[i+6]),cimag(X1[i+6]),creal(X1[i+7]),cimag(X1[i+7]));
				fclose(output);
				break;
				//Two force cases because of the flag
			case(4):	
				output_old = fopen("force_0_old", "w");
				for(int i = 0; i< kmom; i+=4)
					fprintf(output_old, "%f\t%f\t%f\t%f\n", dSdpi[i], dSdpi[i+1], dSdpi[i+2], dSdpi[i+3]);
				fclose(output_old);
				Force(dSdpi, 0, rescgg);	
				output = fopen("force_0", "w");
				for(int i = 0; i< kmom; i+=4)
					fprintf(output, "%f\t%f\t%f\t%f\n", dSdpi[i], dSdpi[i+1], dSdpi[i+2], dSdpi[i+3]);
				fclose(output);
				break;
			case(5):	
				output_old = fopen("force_1_old", "w");
				for(int i = 0; i< kmom; i+=4)
					fprintf(output_old, "%f\t%f\t%f\t%f\n", dSdpi[i], dSdpi[i+1], dSdpi[i+2], dSdpi[i+3]);
				fclose(output_old);
				Force(dSdpi, 1, rescgg);	
				output = fopen("force_1", "w");
				for(int i = 0; i< kmom; i+=4)
					fprintf(output, "%f\t%f\t%f\t%f\n", dSdpi[i], dSdpi[i+1], dSdpi[i+2], dSdpi[i+3]);
				fclose(output);
				break;
			case(6):
				output = fopen("Measure", "w");
				int itercg=0;
				double pbp, endenf, denf; Complex qq, qbqb;
				Measure(&pbp, &endenf, &denf, &qq, &qbqb, respbp, &itercg);
				fprintf(output,"pbp=%f\tendenf=%f\tdenf=%f\nqq=%f+(%f)i\tqbqb=%f+(%f)i\titercg=%i\n\n",
						pbp,endenf,denf,creal(qq),cimag(qq),creal(qbqb),cimag(qbqb),itercg);
				//				Congradp(0,respbp,&itercg);
				for(int i = 0; i< kferm; i+=8)
					fprintf(output, "%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n%f+%fI\t%f+%fI\n\n",
							creal(xi[i]),cimag(xi[i]),creal(xi[i+1]),cimag(xi[i+1]),
							creal(xi[i+2]),cimag(xi[i+2]),creal(xi[i+3]),cimag(xi[i+3]),
							creal(xi[i+4]),cimag(xi[i+4]),creal(xi[i+5]),cimag(xi[i+5]),
							creal(xi[i+6]),cimag(xi[i+6]),creal(xi[i+7]),cimag(xi[i+7])	);
				fclose(output);
				break;
		}
	}

	//George Michael's favourite bit of the code
#ifdef __NVCC__
	hipFree(dk4m); hipFree(dk4p); hipFree(R1); hipFree(dSdpi); hipFree(pp);
	hipFree(Phi); hipFree(u11t); hipFree(u12t); hipFree(xi);
	hipFree(X0); hipFree(X1); hipFree(u11); hipFree(u12);
	hipFree(id); hipFree(iu); hipFree(hd); hipFree(hu);
	hipFree(pcoord);
#elif defined USE_MKL
	mkl_free(dk4m); mkl_free(dk4p); mkl_free(R1); mkl_free(dSdpi); mkl_free(pp);
	mkl_free(Phi); mkl_free(u11t); mkl_free(u12t); mkl_free(xi);
	mkl_free(X0); mkl_free(X1); mkl_free(u11); mkl_free(u12);
	mkl_free(id); mkl_free(iu); mkl_free(hd); mkl_free(hu);
	mkl_free(pcoord);
#else
	free(dk4m); free(dk4p); free(R1); free(dSdpi); free(pp);
	free(Phi); free(u11t); free(u12t); free(xi);
	free(X0); free(X1); free(u11); free(u12);
	free(id); free(iu); free(hd); free(hu);
	free(pcoord);
#endif

	MPI_Finalise();
	exit(0);
}
#endif
