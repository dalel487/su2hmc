#include "hip/hip_runtime.h"
/*
 *Code for fermionic observables
 */
#include	<matrices.h>
#include	<random.h>
#include	<su2hmc.h>
int Measure(double *pbp, double *endenf, double *denf, Complex *qq, Complex *qbqb, double res, int *itercg){
	/*
	 * Calculate fermion expectation values via a noisy estimator
	 * -matrix inversion via conjugate gradient algorithm
	 * solves Mx=x1
	 * (Numerical Recipes section 2.10 pp.70-73)   
	 * uses NEW lookup tables **
	 * Implimented in CongradX
	 *
	 * Calls:
	 * =====
	 * Gauss_z
	 * Par_dsum
	 * ZHalo_swap_dir
	 * DHalo_swap_dir
	 *
	 * Globals:
	 * =======
	 * Phi, X0, xi, R1, u11t, u12t 
	 *
	 * Parameters:
	 * ==========
	 * double *pbp:		Pointer to ψ-bar ψ
	 * double endenf:
	 * double denf:
	 * complex qq:
	 * complex qbqb:
	 * double res:
	 * int itercg:
	 *
	 * Returns:
	 * =======
	 * Zero on success, integer error code otherwise
	 */
	const char *funcname = "Measure";
	//This x is just a storage container

#ifdef __NVCC__
	int device=-1;
	hipGetDevice(&device);
	Complex *x;
	hipMallocManaged(&x,kfermHalo*sizeof(Complex), hipMemAttachGlobal);
#elif defined USE_MKL
	complex *x = mkl_malloc(kfermHalo*sizeof(complex), AVX);
#else
	complex *x = malloc(kfermHalo*sizeof(complex));
#endif
	//Setting up noise. I don't see any reason to loop

	//The root two term comes from the fact we called gauss0 in the fortran code instead of gaussp
#if (defined(USE_RAN2)||!defined(USE_MKL))
	Gauss_z(xi, kferm, 0, 1/sqrt(2));
#else
	vdRngGaussian(VSL_RNG_METHOD_GAUSSIAN_ICDF, stream, 2*kferm, (double*)xi, 0, 1/sqrt(2));
#endif
	hipMemPrefetchAsync(xi, kferm*sizeof(Complex),device,NULL);
	memcpy(x, xi, kferm*sizeof(Complex));

	//R_1= M^† Ξ 
	//R1 is local in fortran but since its going to be reset anyway I'm going to recycle the
	//global
	Dslashd(R1, xi);
	//Copying R1 to the first (zeroth) flavour index of Phi
	//This should be safe with memcpy since the pointer name
	//references the first block of memory for that pointer
	memcpy(Phi, R1, nc*ngorkov*kvol*sizeof(Complex));
	memcpy(xi, R1, nc*ngorkov*kvol*sizeof(Complex));

	//Evaluate xi = (M^† M)^-1 R_1 
	hipMemPrefetchAsync(x, kferm*sizeof(Complex),device,NULL);
	Congradp(0, res, itercg);
#ifdef __NVCC__
	Complex buff;
	hipblasZdotc(cublas_handle,kferm, (hipDoubleComplex *)x, 1, (hipDoubleComplex *)xi,  1, (hipDoubleComplex *)&buff);
	*pbp=buff.real();
#elif (defined USE_MKL || defined USE_BLAS)
	complex buff;
	cblas_zdotc_sub(kferm, x, 1, xi,  1, &buff);
	*pbp=creal(buff);
#else
	*pbp = 0;
#pragma unroll
	for(int i=0;i<kferm;i++)
		*pbp+=creal(conj(x[i])*xi[i]);
#endif
	Par_dsum(pbp);
	*pbp/=4*gvol;

	*qbqb=0; *qq=0;
#if (defined USE_MKL || defined USE_BLAS)
#pragma unroll
	for(int idirac = 0; idirac<ndirac; idirac++){
		int igork=idirac+4;
		//Unrolling the colour indices, Then its just (γ_5*x)*Ξ or (γ_5*Ξ)*x 
#pragma unroll
		for(int ic = 0; ic<nc; ic++){
			Complex dot;
			//Because we have kvol on the outer index and are summing over it, we set the
			//step for BLAS to be ngorkov*nc=16. 
			//Does this make sense to do on the GPU?
#ifdef __NVCC__
			hipblasZdotc(cublas_handle,kferm, (hipDoubleComplex *)&x[idirac*nc+ic], ngorkov*nc,\
			(hipDoubleComplex *)&xi[igork*nc+ic],  ngorkov*nc, (hipDoubleComplex *)&dot);
#elif (defined USE_MKL || defined USE_BLAS)
			cblas_zdotc_sub(kvol, &x[idirac*nc+ic], ngorkov*nc, &xi[igork*nc+ic], ngorkov*nc, &dot);
			*qbqb+=gamval[4][idirac]*dot;
			#endif
#ifdef __NVCC__
			hipblasZdotc(cublas_handle,kferm, (hipDoubleComplex *)&x[igork*nc+ic], ngorkov*nc,\
			(hipDoubleComplex *)&xi[idirac*nc+ic],  ngorkov*nc, (hipDoubleComplex *)&dot);
#elif (defined USE_MKL || defined USE_BLAS)
			cblas_zdotc_sub(kvol, &x[igork*nc+ic], ngorkov*nc, &xi[idirac*nc+ic], ngorkov*nc, &dot);
			*qq-=gamval[4][idirac]*dot;
			#endif
		}
	}
#else
#pragma unroll(2)
	for(int i=0; i<kvol; i++)
		//What is the optimal order to evaluate these in?
		for(int idirac = 0; idirac<ndirac; idirac++){
			int igork=idirac+4;
			*qbqb+=gamval[4][idirac]*conj(x[(i*ngorkov+idirac)*nc])*xi[(i*ngorkov+igork)*nc];
			*qq-=gamval[4][idirac]*conj(x[(i*ngorkov+igork)*nc])*xi[(i*ngorkov+idirac)*nc];
			*qbqb+=gamval[4][idirac]*conj(x[(i*ngorkov+idirac)*nc+1])*xi[(i*ngorkov+igork)*nc+1];
			*qq-=gamval[4][idirac]*conj(x[(i*ngorkov+igork)*nc+1])*xi[(i*ngorkov+idirac)*nc+1];
		}
#endif

	//In the FORTRAN Code dsum was used instead despite qq and qbqb being complex
	Par_zsum(qq); Par_zsum(qbqb);
	*qq=(*qq+*qbqb)/(2.0*gvol);
	Complex xu, xd, xuu, xdd;
	xu=0;xd=0;xuu=0;xdd=0;

	//Halos
	ZHalo_swap_dir(x,16,3,DOWN);		ZHalo_swap_dir(x,16,3,UP);
	//Pesky halo exchange indices again
	//The halo exchange for the trial fields was done already at the end of the trajectory
	//No point doing it again

	//Instead of typing id[i*ndim+3] a lot, we'll just assign them to variables.
	//Idea. One loop instead of two loops but for xuu and xdd just use ngorkov-(igorkov+1) instead
#pragma omp parallel for //reduction(+:xd,xu,xdd,xuu) 
	for(int i = 0; i<kvol; i++){
		int did=id[3+ndim*i];
		int uid=iu[3+ndim*i];
#pragma unroll
#pragma omp simd aligned(u11t:AVX,u12t:AVX,xi:AVX,x:AVX,dk4m:AVX,dk4p:AVX) 
		for(int igorkov=0; igorkov<4; igorkov++){
			int igork1=gamin[3][igorkov];
			//For the C Version I'll try and factorise where possible

			xu+=dk4p[did]*(conj(x[(did*ngorkov+igorkov)*nc])*(\
						u11t[did*ndim+3]*(xi[(i*ngorkov+igork1)*nc]-xi[(i*ngorkov+igorkov)*nc])+\
						u12t[did*ndim+3]*(xi[(i*ngorkov+igork1)*nc+1]-xi[(i*ngorkov+igorkov)*nc+1]) )+\
					conj(x[(did*ngorkov+igorkov)*nc+1])*(\
						conj(u11t[did*ndim+3])*(xi[(i*ngorkov+igork1)*nc+1]-xi[(i*ngorkov+igorkov)*nc+1])+\
						conj(u12t[did*ndim+3])*(xi[(i*ngorkov+igorkov)*nc]-xi[(i*ngorkov+igork1)*nc])));

			xd+=dk4m[i]*(conj(x[(uid*ngorkov+igorkov)*nc])*(\
						conj(u11t[i*ndim+3])*(xi[(i*ngorkov+igork1)*nc]+xi[(i*ngorkov+igorkov)*nc])-\
						u12t[i*ndim+3]*(xi[(i*ngorkov+igork1)*nc+1]+xi[(i*ngorkov+igorkov)*nc+1]) )+\
					conj(x[(uid*ngorkov+igorkov)*nc+1])*(\
						u11t[i*ndim+3]*(xi[(i*ngorkov+igork1)*nc+1]+xi[(i*ngorkov+igorkov)*nc+1])+\
						conj(u12t[i*ndim+3])*(xi[(i*ngorkov+igorkov)*nc]+xi[(i*ngorkov+igork1)*nc]) ) );

			int igorkovPP=igorkov+4;
			int igork1PP=igork1+4;
			xuu-=dk4m[did]*(conj(x[(did*ngorkov+igorkovPP)*nc])*(\
						u11t[did*ndim+3]*(xi[(i*ngorkov+igork1PP)*nc]-xi[(i*ngorkov+igorkovPP)*nc])+\
						u12t[did*ndim+3]*(xi[(i*ngorkov+igork1PP)*nc+1]-xi[(i*ngorkov+igorkovPP)*nc+1]) )+\
					conj(x[(did*ngorkov+igorkovPP)*nc+1])*(\
						conj(u11t[did*ndim+3])*(xi[(i*ngorkov+igork1PP)*nc+1]-xi[(i*ngorkov+igorkovPP)*nc+1])+\
						conj(u12t[did*ndim+3])*(xi[(i*ngorkov+igorkovPP)*nc]-xi[(i*ngorkov+igork1PP)*nc]) ) );

			xdd-=dk4p[i]*(conj(x[(uid*ngorkov+igorkovPP)*nc])*(\
						conj(u11t[i*ndim+3])*(xi[(i*ngorkov+igork1PP)*nc]+xi[(i*ngorkov+igorkovPP)*nc])-\
						u12t[i*ndim+3]*(xi[(i*ngorkov+igork1PP)*nc+1]+xi[(i*ngorkov+igorkovPP)*nc+1]) )+\
					conj(x[(uid*ngorkov+igorkovPP)*nc+1])*(\
						u11t[i*ndim+3]*(xi[(i*ngorkov+igork1PP)*nc+1]+xi[(i*ngorkov+igorkovPP)*nc+1])+\
						conj(u12t[i*ndim+3])*(xi[(i*ngorkov+igorkovPP)*nc]+xi[(i*ngorkov+igork1PP)*nc]) ) );
		}
	}
	*endenf=(xu-xd-xuu+xdd).real();
	*denf=(xu+xd+xuu+xdd).real();

	Par_dsum(endenf); Par_dsum(denf);
	*endenf/=2*gvol; *denf/=2*gvol;
	//Future task. Chiral susceptibility measurements
#ifdef __NVCC__
	hipFree(x);
#elif defined USE_MKL
	mkl_free(x);
#else
	free(x);
#endif
	return 0;
}
