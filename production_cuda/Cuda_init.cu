#include <Complex.h>
#include <hip/hip_runtime.h>
int Cuda_init(){
//From Init()
	hipMallocManaged(&dk4m,(kvol+halo)*sizeof(double),hipMemAttachGlobal);
	hipMallocManaged(&dk4p,(kvol+halo)*sizeof(double),hipMemAttachGlobal);
//Also from Init()	
	hipMallocManaged(&u11,ndim*(kvol+halo)*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&u12,ndim*(kvol+halo)*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&u11t,ndim*(kvol+halo)*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&u12t,ndim*(kvol+halo)*sizeof(complex),hipMemAttachGlobal);

//From just before the main loop
	hipMallocManaged(&R1, kfermHalo*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&xi, kfermHalo*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&Phi, nf*kfermHalo*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&X0, nf*kfermHalo*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&X1, kferm2Halo*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&dSdpi, kmomHalo*sizeof(complex),hipMemAttachGlobal);
	hipMallocManaged(&pp, kmomHalo*sizeof(complex),hipMemAttachGlobal);
}
