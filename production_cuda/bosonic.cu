#include "hip/hip_runtime.h"
/*
 * Code for bosonic observables
 * Basically polyakov loop and Plaquette routines
 */
#include	<par_mpi.h>
#include	<su2hmc.h>
__host__ int SU2plaq(double *hg, double *avplaqs, double *avplaqt){
	/* 
	 * Calculates the gauge action using new (how new?) lookup table
	 * Follows a routine called qedplaq in some QED3 code
	 *
	 * Globals:
	 * =======
	 * 
	 *
	 * Parameters:
	 * ===========
	 * double hg
	 * double avplaqs
	 * double avplaqt
	 *
	 * Returns:
	 * =======
	 * Zero on success, integer error code otherwise
	 */
	const char *funcname = "SU2plaq";
	//Was a halo exchange here but moved it outside
	//	The fortran code used several consecutive loops to get the plaquette
	//	Instead we'll just make the arrays variables and do everything in one loop
	//	Should work since in the fortran Sigma11[i] only depends on i components  for example
	double hgs = 0; double hgt = 0;
	//Since the ν loop doesn't get called for μ=0 we'll start at μ=1
	for(int mu=1;mu<ndim;mu++)
		for(int nu=0;nu<mu;nu++)
			//Don't merge into a single loop. Makes vectorisation easier?
			//Or merge into a single loop and dispense with the a arrays?
			cuSU2plaq<<<dimGrid,dimBlock>>>(mu, nu, &hgs, &hgt);
	Par_dsum(&hgs); Par_dsum(&hgt);
	*avplaqs=-hgs/(3.0*gvol); *avplaqt=-hgt/(gvol*3.0);
	*hg=(hgs+hgt)*beta;
#ifdef _DEBUG
	if(!rank)
		printf("hgs=%e  hgt=%e  hg=%e\n", hgs, hgt, *hg);
#endif
	return 0;
}
__host__ double Polyakov(){
	/*
	 * Calculate the Polyakov loop (no prizes for guessing that one...)
	 *
	 * Globals:
	 * =======
	 * u11t, u12t, u11t, u12t
	 *
	 * Calls:
	 * ======
	 * Par_tmul, Par_dsum
	 * 
	 * Parameters:
	 * ==========
	 * double *poly The Polyakov Loop value
	 * 
	 * Returns:
	 * =======
	 * Double corresponding to the polyakov loop
	 */
	const char *funcname = "Polyakov";
	double poly = 0;
	//Originally at the very end before Par_dsum
	//Now all cores have the value for the complete Polyakov line at all spacial sites
	//We need to globally sum over spacial processores but not across time as these
	//are duplicates. So we zero the value for all but t=0
	//This is (according to the FORTRAN code) a bit of a hack
	//I will expand on this hack and completely avoid any work
	//for this case rather than calculating everything just to set it to zero
	int device=-1;
	hipGetDevice(&device);
	Complex *Sigma11,*Sigma12;
	hipMallocManaged(&Sigma11,kvol3*sizeof(Complex),hipMemAttachGlobal);
	//hipMemAdvise(Sigma11,kvol3*sizeof(Complex),hipMemAdviseSetPreferredLocation,device);
	//Sigma12 only used on device unless npt>1. So worth considering device-only memory
#if (npt>1)
	hipMallocManaged(&Sigma12,kvol3*sizeof(Complex),hipMemAttachGlobal);
#else
	hipMalloc(&Sigma12,kvol3*sizeof(Complex));
#endif
	//hipMemAdvise(Sigma12,kvol3*sizeof(Complex),hipMemAdviseSetPreferredLocation,device);
#ifdef __NVCC__
	hipblasZcopy(cublas_handle,kvol3, (hipDoubleComplex *)&u11t[3], ndim, (hipDoubleComplex *)Sigma11, 1);
	hipblasZcopy(cublas_handle,kvol3, (hipDoubleComplex *)&u12t[3], ndim, (hipDoubleComplex *)Sigma12, 1);
#elif (defined USE_MKL || defined USE_BLAS)
	cblas_zcopy(kvol3, &u11t[3], ndim, Sigma11, 1);
	cblas_zcopy(kvol3, &u12t[3], ndim, Sigma12, 1);
#else
	for(int i=0; i<kvol3; i++){
		Sigma11[i]=u11t[i*ndim+3];
		Sigma12[i]=u12t[i*ndim+3];
	}
#endif
	//	Some Fortran commentary
	//	Changed this routine.
	//	u11t and u12t now defined as normal ie (kvol+halo,4).
	//	Copy of Sigma11 and Sigma12 is changed so that it copies
	//	in blocks of ksizet.
	//	Variable indexu also used to select correct element of u11t and u12t 
	//	in loop 10 below.
	//
	//	Change the order of multiplication so that it can
	//	be done in parallel. Start at t=1 and go up to t=T:
	//	previously started at t+T and looped back to 1, 2, ... T-1
	//Buffers
	hipMemPrefetchAsync(Sigma11,kvol3*sizeof(Complex),device,NULL);
	hipMemPrefetchAsync(Sigma12,kvol3*sizeof(Complex),device,NULL);
	cuPolyakov<<<dimGrid,dimBlock>>>(Sigma11,Sigma12);
	//Multiply this partial loop with the contributions of the other cores in the
	//timelike dimension
	//Par_tmul does nothing if there is only a single processor in the time direction. So we only compile
	//its call if it is required
#if (npt>1)
	hipMemPrefetchAsync(Sigma11,kvol3*sizeof(Complex),hipCpuDeviceId,NULL);
	hipMemPrefetchAsync(Sigma12,kvol3*sizeof(Complex),hipCpuDeviceId,NULL);
#ifdef _DEBUG
	printf("Multiplying with MPI\n");
#endif
	Par_tmul(Sigma11, Sigma12);
#endif
#pragma omp parallel for simd reduction(+:poly) aligned(Sigma11:AVX)
	//TODO:	CUDA Reduction
	for(int i=0;i<kvol3;i++)
		poly+=Sigma11[i].real();
	hipFree(Sigma11); hipFree(Sigma12);

	if(pcoord[3+rank*ndim]) poly = 0;
	Par_dsum(&poly);
	poly/=gvol3;
	return poly;	
}
//CUDA Kernels
__global__ void cuSU2plaq(int mu, int nu, double *hgs, double *hgt){
	char *funcname = "cuSU2plaq";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize){
		//Save us from typing iu[mu+ndim*i] everywhere
		int uidm = iu[mu+ndim*i]; 

		Complex Sigma11=u11t[i*ndim+mu]*u11t[uidm*ndim+nu]-u12t[i*ndim+mu]*conj(u12t[uidm*ndim+nu]);
		Complex Sigma12=u11t[i*ndim+mu]*u12t[uidm*ndim+nu]+u12t[i*ndim+mu]*conj(u11t[uidm*ndim+nu]);

		int uidn = iu[nu+ndim*i]; 
		Complex a11=Sigma11*conj(u11t[uidn*ndim+mu])+Sigma12*conj(u12t[uidn*ndim+mu]);
		Complex a12=-Sigma11*u12t[uidn*ndim+mu]+Sigma12*u11t[uidn*ndim+mu];

		Sigma11=a11*conj(u11t[i*ndim+nu])+a12*conj(u12t[i*ndim+nu]);
		//				Sigma12[i]=-a11[i]*u12t[i*ndim+nu]+a12*u11t[i*ndim+mu];
		//				Not needed in final result as it traces out

		switch(mu){
			//Time component
			case(ndim-1):	atomicAdd(hgt, -Sigma11.real());
						break;
						//Space component
			default:	atomicAdd(hgs, -Sigma11.real());
					break;
		}
	}
}
__global__ void cuPolyakov(Complex *Sigma11, Complex * Sigma12){
	char * funcname = "cuPolyakov";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
#pragma unroll
	for(int it=1;it<ksizet;it++)
		for(int i=threadId;i<kvol3;i+=gsize){
			//Seems a bit more efficient to increment indexu instead of reassigning
			//it every single loop
			int indexu=it*kvol3+i;
			Complex a11=Sigma11[i]*u11t[indexu*ndim+3]-Sigma12[i]*conj(u12t[indexu*ndim+3]);
			//Instead of having to store a second buffer just assign it directly
			Sigma12[i]=Sigma11[i]*u12t[indexu*ndim+3]+Sigma12[i]*conj(u11t[indexu*ndim+3]);
			Sigma11[i]=a11;
		}
}
