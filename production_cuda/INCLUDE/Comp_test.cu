#include "hip/hip_runtime.h"
#include <cuda_complex.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define complexd complex<double>
#define complexf complex<float>
__global__ void test_run(){
	complexd z1={1.0/(1+threadIdx.x+blockIdx.z),1.0*threadIdx.x+1+blockIdx.y};
	complexd z2={1.0/(1+threadIdx.y+blockIdx.z),1.0*threadIdx.y+1+blockIdx.x};
	complexd z3 = z1+z2;
	complexd z4 = z1 -z2;
	complexd z5 = conj(z2);
	complexd z6 = z1*z2;
	double x = norm(z1);
	complexd z8 = z1/z2;
	bool eq = (z3==z4);
	bool neq = (z3!=z4);
	
	double x1 = creal(z1);
	double x2 = cimag(z2);
	complexd z9 = x1+z1;
	complexd z10 = z1+x1;
	complexd z11 =x2-z2;
	complexd z12 = z2-x2;
	complexd z13 = x1*z1;
	complexd z14 = z1*x1;
	complexd z15 = x2/z2;
	complexd z16 = z2/x2;
	bool deq1 = (x1==z1);
	bool dneq1 = (x1!=z1);
	bool deq2 = (z2==x2);
	bool dneq2 = (z2!=x2);
	
	printf("For z1=%f+%fi and z2=%f+%fi, z13=creal(z1)*z2=%f+%fi\n",
	creal(z1),cimag(z1),creal(z2),cimag(z2),creal(z13),cimag(z13));
}
int main(int argc, char *argv[]){
	dim3 dimGrid(1024,1024,1024);
	dim3 dimBlock(16,16,2);
	test_run<<<dimGrid,dimBlock>>>();
	return 0;
}
