#include "hip/hip_runtime.h"
/*
 * Code for force calculations.
 * Requires multiply.cu to work
 */
#include	<matrices.h>
#include	<par_mpi.h>
#include	<su2hmc.h>
//Calling functions
void cuGauge_force(int mu, Complex_f *Sigma11, Complex_f *Sigma12, Complex_f *u11t,Complex_f *u12t,double *dSdpi,float beta,\
		dim3 dimGrid, dim3 dimBlock){
	const char *funcname = "Gauge_force";
	cuGaugeForce<<<dimGrid,dimBlock>>>(mu,Sigma11,Sigma12,dSdpi,u11t,u12t,beta);
}
void cuPlus_staple(int mu, int nu, unsigned int *iu, Complex_f *Sigma11, Complex_f *Sigma12, Complex_f *u11t, Complex_f *u12t,\
		dim3 dimGrid, dim3 dimBlock){
	const char *funcname="Plus_staple";
	Plus_staple<<<dimGrid,dimBlock>>>(mu, nu, iu, Sigma11, Sigma12,u11t,u12t);
}
void cuMinus_staple(int mu, int nu, unsigned int *iu, unsigned int *id, Complex_f *Sigma11, Complex_f *Sigma12,\
		Complex_f *u11sh, Complex_f *u12sh,Complex_f *u11t, Complex_f *u12t,dim3 dimGrid, dim3 dimBlock){
	const char *funcname="Minus_staple";
	Minus_staple<<<dimGrid,dimBlock>>>(mu, nu, iu, id,Sigma11,Sigma12,u11sh,u12sh,u11t,u12t);
}
void cuForce(double *dSdpi, Complex_f *u11t, Complex_f *u12t, Complex_f *X1, Complex_f *X2, \
		Complex_f *gamval,float *dk4m, float *dk4p,unsigned int *iu,int *gamin,\
		float akappa, dim3 dimGrid, dim3 dimBlock){
	const char *funcname = "Force";
	//X1=(M†M)^{1} Phi
	Transpose_U(iu,ndim,kvol,dimGrid,dimBlock);
	Transpose_d(dSdpi,nadj*ndim,kvol,dimGrid,dimBlock);
	Transpose_c(u11t,ndim,kvol,dimGrid,dimBlock); Transpose_c(u12t,ndim,kvol,dimGrid,dimBlock);
//	Transpose_z(X1,ndirac*nc,kvol,dimGrid,dimBlock); Transpose_z(X2,ndirac*nc,kvol,dimGrid,dimBlock);
	cudaDeviceSynchronise();
	for(int mu=0;mu<3;mu++){
		cuForce_s<<<dimGrid,dimBlock,0,streams[mu]>>>(dSdpi,u11t,u12t,X1,X2,gamval,iu,gamin,akappa,mu);
		//			cuForce_s1<<<dimGrid,dimBlock,0,streams[mu*nadj+1]>>>(dSdpi,u11t,u12t,X1,X2,gamval,dk4m,dk4p,iu,gamin,akappa,idirac,mu);
		//			cuForce_s2<<<dimGrid,dimBlock,0,streams[mu*nadj+2]>>>(dSdpi,u11t,u12t,X1,X2,gamval,dk4m,dk4p,iu,gamin,akappa,idirac,mu);
	}
	//Set stream for time direction
	int mu=3;
	cuForce_t<<<dimGrid,dimBlock,0,streams[mu]>>>(dSdpi,u11t,u12t,X1,X2,gamval,dk4m,dk4p,iu,gamin,akappa);
	cudaDeviceSynchronise();
//	Transpose_z(X1,kvol,ndirac*nc,dimGrid,dimBlock); Transpose_z(X2,kvol,ndirac*nc,dimGrid,dimBlock);
	Transpose_c(u11t,kvol,ndim,dimGrid,dimBlock); Transpose_c(u12t,kvol,ndim,dimGrid,dimBlock);
	Transpose_U(iu,kvol,ndim,dimGrid,dimBlock); 	
	Transpose_d(dSdpi,kvol,nadj*ndim,dimGrid,dimBlock);
	cudaDeviceSynchronise();
}

//CUDA Kernels
//TODO: Split cuForce into seperateable streams. Twelve in total I Believe?
//A stream for each nadj index,dirac index and each μ (ndim) value
//3*4*4=36 streams total... Pass dirac and μ spatial indices as arguments
__global__ void Plus_staple(int mu, int nu,unsigned int *iu, Complex_f *Sigma11, Complex_f *Sigma12, Complex_f *u11t, Complex_f *u12t){
	const char *funcname = "Plus_staple";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize*bsize){
		int uidm = iu[mu+ndim*i];
		int uidn = iu[nu+ndim*i];
		Complex_f	a11=u11t[uidm*ndim+nu]*conj(u11t[uidn*ndim+mu])+\
							 u12t[uidm*ndim+nu]*conj(u12t[uidn*ndim+mu]);
		Complex_f	a12=-u11t[uidm*ndim+nu]*u12t[uidn*ndim+mu]+\
							 u12t[uidm*ndim+nu]*u11t[uidn*ndim+mu];
		Sigma11[i]+=a11*conj(u11t[i*ndim+nu])+a12*conj(u12t[i*ndim+nu]);
		Sigma12[i]+=-a11*u12t[i*ndim+nu]+a12*u11t[i*ndim+nu];
	}
}
__global__ void Minus_staple(int mu,int nu,unsigned int *iu,unsigned int *id, Complex_f *Sigma11, Complex_f *Sigma12,\
		Complex_f *u11sh, Complex_f *u12sh, Complex_f *u11t, Complex_f *u12t){
	const char *funcname = "Minus_staple";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize*bsize){
		int uidm = iu[mu+ndim*i];
		int didn = id[nu+ndim*i];
		//uidm is correct here
		Complex_f a11=conj(u11sh[uidm])*conj(u11t[didn*ndim+mu])-\
						  u12sh[uidm]*conj(u12t[didn*ndim+mu]);
		Complex_f a12=-conj(u11sh[uidm])*u12t[didn*ndim+mu]-\
						  u12sh[uidm]*u11t[didn*ndim+mu];
		Sigma11[i]+=a11*u11t[didn*ndim+nu]-a12*conj(u12t[didn*ndim+nu]);
		Sigma12[i]+=a11*u12t[didn*ndim+nu]+a12*conj(u11t[didn*ndim+nu]);
	}
}
__global__ void cuGaugeForce(int mu, Complex_f *Sigma11, Complex_f *Sigma12,double* dSdpi,Complex_f *u11t, Complex_f *u12t, float beta){
	const char *funcname = "cuGaugeForce";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	for(int i=threadId;i<kvol;i+=gsize*bsize){
		Complex_f a11 = u11t[i*ndim+mu]*Sigma12[i]+u12t[i*ndim+mu]*conj(Sigma11[i]);
		Complex_f a12 = u11t[i*ndim+mu]*Sigma11[i]+conj(u12t[i*ndim+mu])*Sigma12[i];
		//Not worth splitting into different streams, before we get ideas...
		dSdpi[(i*nadj)*ndim+mu]=beta*a11.imag();
		dSdpi[(i*nadj+1)*ndim+mu]=beta*a11.real();
		dSdpi[(i*nadj+2)*ndim+mu]=beta*a12.imag();
	}
}

__global__ void cuForce_s(double *dSdpi, Complex_f *u11t, Complex_f *u12t, Complex_f *X1, Complex_f *X2, Complex_f *gamval,\
		unsigned int *iu, int *gamin,float akappa, int mu){
	const char *funcname = "cuForce";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int bthreadId= (threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	const int gthreadId= blockId * bsize+bthreadId;
	for(int i=gthreadId;i<kvol;i+=gsize*bsize){
		//Complex_f u11s=u11t[i*ndim+mu];	Complex_f u12s=u12t[i*ndim+mu];
		const Complex_f u11s=u11t[i+kvol*mu];	const Complex_f u12s=u12t[i+kvol*mu];
		//const int uid = iu[mu+ndim*i];
		const int uid = iu[mu*kvol+i];
		for(int idirac=0;idirac<ndirac;idirac++){
			Complex_f X1s[nc];	 Complex_f X1su[nc];
			Complex_f X2s[nc];	 Complex_f X2su[nc];
			//	phi[(i*ndirac+idirac)*nc]+=
			//phi_s[idirac*nc+c]=phi[i+kvol*(c+nc*idirac)];
			//X1s[0]=X1[(i*ndirac+idirac)*nc];	X1s[1]=X1[(i*ndirac+idirac)*nc+1];
			//X1su[0]=X1[(uid*ndirac+idirac)*nc];	X1su[1]=X1[(uid*ndirac+idirac)*nc+1];
			//X2s[0]=X2[(i*ndirac+idirac)*nc];	X2s[1]=X2[(i*ndirac+idirac)*nc+1];
			//X2su[0]=X2[(uid*ndirac+idirac)*nc];	X2su[1]=X2[(uid*ndirac+idirac)*nc+1];
			X1s[0]=X1[i+kvol*(nc*idirac)]; X1s[1]=X1[i+kvol*(1+nc*idirac)];
			X1su[0]=X1[uid+kvol*(nc*idirac)]; X1su[1]=X1[uid+kvol*(1+nc*idirac)];
			X2s[0]=X2[i+kvol*(nc*idirac)]; X2s[1]=X2[i+kvol*(1+nc*idirac)];
			X2su[0]=X2[uid+kvol*(nc*idirac)]; X2su[1]=X2[uid+kvol*(1+nc*idirac)];

			float dSdpis[3];
			//dSdpis[0]=dSdpi[(i*nadj)*ndim+mu];
			dSdpis[0]=dSdpi[i+kvol*(mu)];
			//Multiplying by i and taking the real component is the same as taking the negative imaginary component
			dSdpis[0]+=-akappa*(
					conj(X1s[0])*(-conj(u12s)*X2su[0]+conj(u11s)*X2su[1])
					+conj(X1su[0])*(u12s*X2s[0]-conj(u11s)*X2s[1])
					+conj(X1s[1])*(u11s*X2su[0]+u12s*X2su[1])
					+conj(X1su[1])*(-u11s*X2s[0]-conj(u12s)*X2s[1])).imag();

			//dSdpis[1]=dSdpi[(i*nadj+1)*ndim+mu];
			dSdpis[1]=dSdpi[i+kvol*(ndim+mu)];
			dSdpis[1]+=akappa*(
					(conj(X1s[0])* (-conj(u12s)*X2su[0] +conj(u11s)*X2su[1])
					 +conj(X1su[0])* (-u12s *X2s[0] -conj(u11s)*X2s[1])
					 +conj(X1s[1])* (-u11s *X2su[0]-u12s *X2su[1])
					 +conj(X1su[1])* (u11s *X2s[0]-conj(u12s)*X2s[1]))).real();

			//dSdpis[2]=dSdpi[(i*nadj+2)*ndim+mu];
			dSdpis[2]=dSdpi[i+kvol*(2*ndim+mu)];
			dSdpis[2]+=-akappa*(
					conj(X1s[0])*(u11s *X2su[0]+u12s *X2su[1])
					+conj(X1su[0])*(-conj(u11s)*X2s[0]-u12s *X2s[1])
					+conj(X1s[1])*(conj(u12s)*X2su[0]-conj(u11s)*X2su[1])
					+conj(X1su[1])*(-conj(u12s)*X2s[0]+u11s *X2s[1])).imag();

			const int igork1 = gamin[mu*ndirac+idirac];	
			//X2s[0]=X2[(i*ndirac+igork1)*nc];	X2s[1]=X2[(i*ndirac+igork1)*nc+1];
			//X2su[0]=X2[(uid*ndirac+igork1)*nc];	X2su[1]=X2[(uid*ndirac+igork1)*nc+1];
			X2s[0]=X2[i+kvol*(nc*igork1)]; X2s[1]=X2[i+kvol*(1+nc*igork1)];
			X2su[0]=X2[uid+kvol*(nc*igork1)]; X2su[1]=X2[uid+kvol*(1+nc*igork1)];

			dSdpis[0]+=-(gamval[mu*ndirac+idirac]*
					(conj(X1s[0])* (-conj(u12s)*X2su[0]+conj(u11s)*X2su[1])
					 +conj(X1su[0])* (-u12s *X2s[0] +conj(u11s)*X2s[1])
					 +conj(X1s[1])* (u11s *X2su[0]+u12s *X2su[1])
					 +conj(X1su[1])*(u11s *X2s[0] +conj(u12s)*X2s[1]))).imag();
			//dSdpi[(i*nadj)*ndim+mu]=dSdpis[0];
			dSdpi[i+kvol*(mu)]=dSdpis[0];

			dSdpis[1]+=(gamval[mu*ndirac+idirac]*
					(conj(X1s[0])* (-conj(u12s)*X2su[0] +conj(u11s)*X2su[1])
					 +conj(X1su[0])* (u12s *X2s[0]+conj(u11s)*X2s[1])
					 +conj(X1s[1])*(-u11s *X2su[0]-u12s *X2su[1])
					 +conj(X1su[1])* (-u11s *X2s[0]+conj(u12s)*X2s[1]))).real();
			//dSdpi[(i*nadj+1)*ndim+mu]=dSdpis[1];
			dSdpi[i+kvol*(ndim+mu)]=dSdpis[1];

			dSdpis[2]+=-(gamval[mu*ndirac+idirac]*
					(conj(X1s[0])*(u11s *X2su[0]+u12s *X2su[1])
					 +conj(X1su[0])*(conj(u11s)*X2s[0]+u12s *X2s[1])
					 +conj(X1s[1])*(conj(u12s)*X2su[0]-conj(u11s)*X2su[1])
					 +conj(X1su[1])*(conj(u12s)*X2s[0]-u11s *X2s[1]))).imag();
			//dSdpi[(i*nadj+2)*ndim+mu]=dSdpis[2];
			dSdpi[i+kvol*(2*ndim+mu)]=dSdpis[2];
		}
	}
}
__global__ void cuForce_t(double *dSdpi, Complex_f *u11t, Complex_f *u12t, Complex_f *X1, Complex_f *X2, Complex_f *gamval,\
		float *dk4m, float *dk4p, unsigned int *iu, int *gamin,float akappa){
	const char *funcname = "cuForce";
	//Up indices
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int bthreadId= (threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	const int gthreadId= blockId * bsize+bthreadId;
	const int mu=3;
	for(int i=gthreadId;i<kvol;i+=gsize*bsize){
		//Up indices
		//const int uid = iu[mu+ndim*i];
		const int uid = iu[mu*kvol+i];
		//	Complex_f u11s=u11t[i*ndim+mu];	Complex_f u12s=u12t[i*ndim+mu];
		const Complex_f u11s=u11t[i+kvol*mu];	const Complex_f u12s=u12t[i+kvol*mu];
		const float dk4ms=dk4m[i];	const float dk4ps=dk4p[i];

		for(int idirac=0;idirac<ndirac;idirac++){
			Complex X1s[nc];	 Complex X1su[nc];
			Complex_f X2s[nc];	 Complex_f X2su[nc];
			//X1s[0]=X1[(i*ndirac+idirac)*nc];	X1s[1]=X1[(i*ndirac+idirac)*nc+1];
			//X1su[0]=X1[(uid*ndirac+idirac)*nc];	X1su[1]=X1[(uid*ndirac+idirac)*nc+1];
			//X2s[0]=X2[(i*ndirac+idirac)*nc];	X2s[1]=X2[(i*ndirac+idirac)*nc+1];
			//X2su[0]=X2[(uid*ndirac+idirac)*nc];	X2su[1]=X2[(uid*ndirac+idirac)*nc+1];
			X1s[0]=X1[i+kvol*(nc*idirac)]; X1s[1]=X1[i+kvol*(1+nc*idirac)];
			X1su[0]=X1[uid+kvol*(nc*idirac)]; X1su[1]=X1[uid+kvol*(1+nc*idirac)];
			X2s[0]=X2[i+kvol*(nc*idirac)]; X2s[1]=X2[i+kvol*(1+nc*idirac)];
			X2su[0]=X2[uid+kvol*(nc*idirac)]; X2su[1]=X2[uid+kvol*(1+nc*idirac)];

			float dSdpis[3];
		//	dSdpis[0]=dSdpi[(i*nadj)*ndim+mu];
			dSdpis[0]=dSdpi[i+kvol*(mu)];
			dSdpis[0]+=-(conj(X1s[0])* (dk4ms*(-conj(u12s)*X2su[0]+conj(u11s)*X2su[1]))
					 +conj(X1su[0])* (dk4ps*(+u12s *X2s[0]-conj(u11s)*X2s[1]))
					 +conj(X1s[1])*(dk4ms*(u11s *X2su[0]+u12s *X2su[1]))
					 +conj(X1su[1])*(dk4ps*(-u11s *X2s[0]-conj(u12s)*X2s[1]))).imag();
		//	dSdpis[1]=dSdpi[(i*nadj+1)*ndim+mu];
			dSdpis[1]=dSdpi[i+kvol*(ndim+mu)];
			dSdpis[1]+=(conj(X1s[0])*(dk4ms*(-conj(u12s)*X2su[0]+conj(u11s)*X2su[1]))
					+conj(X1su[0])*(dk4ps*(-u12s *X2s[0]-conj(u11s)*X2s[1]))
					+conj(X1s[1])*(dk4ms*(-u11s *X2su[0]-u12s *X2su[1]))
					+conj(X1su[1])*(dk4ps*( u11s *X2s[0]-conj(u12s)*X2s[1]))).real();

			//dSdpis[2]=dSdpi[(i*nadj+2)*ndim+mu];
			dSdpis[2]=dSdpi[i+kvol*(2*ndim+mu)];
			dSdpis[2]+=-(conj(X1s[0])* (dk4ms* (u11s *X2su[0]+u12s *X2su[1]))
					 +conj(X1su[0])*(dk4ps*(-conj(u11s)*X2s[0]-u12s *X2s[1]))
					 +conj(X1s[1])* (dk4ms* (conj(u12s)*X2su[0]-conj(u11s)*X2su[1]))
					 +conj(X1su[1])* (dk4ps*(-conj(u12s)*X2s[0]+u11s *X2s[1]))).imag();

			const int igork1 = gamin[mu*ndirac+idirac];	
			//X2s[0]=X2[(i*ndirac+igork1)*nc];	X2s[1]=X2[(i*ndirac+igork1)*nc+1];
			//X2su[0]=X2[(uid*ndirac+igork1)*nc];	X2su[1]=X2[(uid*ndirac+igork1)*nc+1];
			X2s[0]=X2[i+kvol*(nc*igork1)]; X2s[1]=X2[i+kvol*(1+nc*igork1)];
			X2su[0]=X2[uid+kvol*(nc*igork1)]; X2su[1]=X2[uid+kvol*(1+nc*igork1)];

			dSdpis[0]+=-(conj(X1s[0])*(dk4ms*(-conj(u12s)*X2su[0]+conj(u11s)*X2su[1]))
					 +conj(X1su[0])* (-dk4ps*(u12s *X2s[0]-conj(u11s)*X2s[1]))
					 +conj(X1s[1])*(dk4ms*(u11s *X2su[0]+u12s *X2su[1]))
					 +conj(X1su[1])*(-dk4ps*(-u11s *X2s[0]-conj(u12s)*X2s[1]))).imag();
			//dSdpi[(i*nadj)*ndim+mu]=dSdpis[0];
			dSdpi[i+kvol*(mu)]=dSdpis[0];

			dSdpis[1]+=(
					(conj(X1s[0])*(dk4ms*(-conj(u12s)*X2su[0]+conj(u11s)*X2su[1]))
					 +conj(X1su[0])*(-dk4ps*(-u12s *X2s[0]-conj(u11s)*X2s[1]))
					 +conj(X1s[1])*(dk4ms*(-u11s*X2su[0]-u12s *X2su[1]))
					 +conj(X1su[1])*(-dk4ps*(u11s*X2s[0]-conj(u12s)*X2s[1])))).real();
			//dSdpi[(i*nadj+1)*ndim+mu]=dSdpis[1];
			dSdpi[i+kvol*(ndim+mu)]=dSdpis[1];

			dSdpis[2]+=-(conj(X1s[0])*(dk4ms*(u11s*X2su[0] +u12s *X2su[1]))
					 +conj(X1su[0])*(-dk4ps*(-conj(u11s)*X2s[0]-u12s *X2s[1]))
					 +conj(X1s[1])*(dk4ms* (conj(u12s)*X2su[0]-conj(u11s)*X2su[1]))
					 +conj(X1su[1])*(-dk4ps*(-conj(u12s)*X2s[0]+u11s *X2s[1]))).imag();
			//dSdpi[(i*nadj+2)*ndim+mu]=dSdpis[2];
			dSdpi[i+kvol*(2*ndim+mu)]=dSdpis[2];
		}
	}
}
