#include <hip/hip_runtime.h>
#include <complex>
#include <par_mpi.h>
#include <su2hmc.h>
__host__ int Cuda_init(){
//From Init()
	hipMallocManaged(&dk4m,(kvol+halo)*sizeof(double),hipMemAttachGlobal);
	hipMallocManaged(&dk4p,(kvol+halo)*sizeof(double),hipMemAttachGlobal);
//Also from Init()	
	hipMallocManaged(&u11,ndim*(kvol+halo)*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&u12,ndim*(kvol+halo)*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&u11t,ndim*(kvol+halo)*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&u12t,ndim*(kvol+halo)*sizeof(complex<double>),hipMemAttachGlobal);

//From just before the main loop
	hipMallocManaged(&R1, kfermHalo*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&xi, kfermHalo*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&Phi, nf*kfermHalo*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&X0, nf*kfermHalo*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&X1, kferm2Halo*sizeof(complex<double>),hipMemAttachGlobal);
	hipMallocManaged(&pp, kmomHalo*sizeof(complex<double>),hipMemAttachGlobal);
	return 0;
}
