#include "hip/hip_runtime.h"
/*
 * Code for bosonic observables
 * Basically polyakov loop and Plaquette routines
 */
#include	<par_mpi.h>
#include	<su2hmc.h>
void cuAverage_Plaquette(double *hgs, double *hgt, Complex *u11t, Complex *u12t, unsigned int *iu,dim3 dimGrid, dim3 dimBlock){
	double *hgs_d, *hgt_d;
	hipMalloc(&hgs_d,kvol*sizeof(double));
	hipMemset(hgs_d,0,kvol*sizeof(double));
	hipMalloc(&hgt_d,kvol*sizeof(double));
	hipMemset(hgt_d,0,kvol*sizeof(double));

	cuAverage_Plaquette<<<dimGrid,dimBlock>>>(hgs, hgt, u11t, u12t, iu);
	*hgs= thrust::reduce(thrust::host,hgs_d,hgt_d+kvol);
	*hgt= thrust::reduce(thrust::host,hgt_d,hgt_d+kvol);
	hipFree(hgs_d); hipFree(hgt_d);
}
void cuPolyakov(Complex *Sigma11, Complex * Sigma12, Complex *u11t, Complex *u12t, dim3 dimGrid, dim3 dimBlock){
	cuPolyakov<<<dimGrid,dimBlock>>>(Sigma11,Sigma12,u11t,u12t);
}
//CUDA Kernels
__global__ void cuAverage_Plaquette(double *hgs_d, double *hgt_d, Complex *u11t, Complex *u12t, unsigned int *iu){
	char *funcname = "cuSU2plaq";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
	//TODO: Chck if μ and ν loops inside of site loop is faster. I suspect it is due to memory locality.
	for(int i=threadId;i<kvol;i+=bsize*gsize)
		for(int mu=1;mu<ndim;mu++)
			for(int nu=0;nu<mu;nu++){
				//Save us from typing iu[mu+ndim*i] everywhere
				//This is threadsafe as the μ and ν loops are not distributed across threads
				switch(mu){
					//Time component
					case(ndim-1):	hgt_d[i]-= -SU2plaq(u11t,u12t,iu,i,mu,nu);
										break;
										//Space component
					default:	hgs_d[i] -=SU2plaq(u11t,u12t,iu,i,mu,nu);
								break;
				}
			}
}
__device__ inline double SU2plaq(Complex *u11t, Complex *u12t, unsigned int *iu, int i, int mu, int nu){
	/*
	 * Calculates the plaquette at site i in the μ-ν direction
	 *
	 * Parameters:
	 * ==========
	 * Complex u11t, u12t:	Trial fields
	 * unsignedi int *iu:	Upper halo indices
	 * int mu, nu:				Plaquette direction. Note that mu and nu can be negative
	 * 							to facilitate calculating plaquettes for Clover terms. No
	 * 							sanity checks are conducted on them in this routine.
	 *
	 * Returns:
	 * ========
	 * double corresponding to the plaquette value
	 *
	 */
	const char *funcname = "SU2plaq";
	int uidm = iu[mu+ndim*i]; 

	Complex Sigma11=u11t[i*ndim+mu]*u11t[uidm*ndim+nu]-u12t[i*ndim+mu]*conj(u12t[uidm*ndim+nu]);
	Complex Sigma12=u11t[i*ndim+mu]*u12t[uidm*ndim+nu]+u12t[i*ndim+mu]*conj(u11t[uidm*ndim+nu]);

	int uidn = iu[nu+ndim*i]; 
	Complex a11=Sigma11*conj(u11t[uidn*ndim+mu])+Sigma12*conj(u12t[uidn*ndim+mu]);
	Complex a12=-Sigma11*u12t[uidn*ndim+mu]+Sigma12*u11t[uidn*ndim+mu];

	Sigma11=a11*conj(u11t[i*ndim+nu])+a12*conj(u12t[i*ndim+nu]);
	//				Sigma12[i]=-a11[i]*u12t[i*ndim+nu]+a12*u11t[i*ndim+mu];
	//				Not needed in final result as it traces out
	return Sigma11.real();
}
__global__ void cuPolyakov(Complex *Sigma11, Complex * Sigma12, Complex * u11t,Complex *u12t){
	char * funcname = "cuPolyakov";
	const int gsize = gridDim.x*gridDim.y*gridDim.z;
	const int bsize = blockDim.x*blockDim.y*blockDim.z;
	const int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
	const int threadId= blockId * bsize+(threadIdx.z * blockDim.y+ threadIdx.y)* blockDim.x+ threadIdx.x;
#pragma unroll
	for(int it=1;it<ksizet;it++)
		for(int i=threadId;i<kvol3;i+=gsize){
			//Seems a bit more efficient to increment indexu instead of reassigning
			//it every single loop
			int indexu=it*kvol3+i;
			Complex a11=Sigma11[i]*u11t[indexu*ndim+3]-Sigma12[i]*conj(u12t[indexu*ndim+3]);
			//Instead of having to store a second buffer just assign it directly
			Sigma12[i]=Sigma11[i]*u12t[indexu*ndim+3]+Sigma12[i]*conj(u11t[indexu*ndim+3]);
			Sigma11[i]=a11;
		}
}
